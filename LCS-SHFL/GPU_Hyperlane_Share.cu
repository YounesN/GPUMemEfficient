#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

//#define ALL
//#define DEBUG
//#define DEBUG1
//#define DEBUG2
//#define DEBUG3

using namespace std;
__device__ int row = 0;

__device__ void moveToSharePrev(volatile int *table, volatile int *dev_table, int tileAddress, int warp, int thread, int idx, int tileX, int tileY, int rowsize, int segLengthX, int segLengthY, int warpbatch, int warpsize){
	//potential bank conflict for accessing the data of each anti-diagonal
	//to avoid bank conflict, have to adjust the memory layout by introducing dummy elements.
	//padding elements can be used as the dummy elements, but the number of padding of each dimension has to be an odd number.
/*
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			table[idx] = dev_table[pos];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
*/
	int glbpos = tileAddress + (rowsize - 1) + warp * (rowsize - 1) + idx;
	int shrpos = warp * warpsize + idx;
	if (thread < tileX)
		upperbuff[thread] = dev_table[tileAddress + paddX - 1 + thread];
	for (; warp < segLengthY; warp+=warpbatch){
		table[shrpos] = dev_table[glbpos];
		shrpos += (warpbatch * warpsize);
		glbpos += (warpbatch * (rowsize - 1) );
	}

}

__device__ void moveToShareUpper(volatile int *upperbuff, volatile int *dev_table, int tileAddress, int idx, int tileX, int offset){
		upperbuff[idx] = dev_table[tileAddress + paddX - 1 + offset + (rowsize - 1) * warp * warpsize + idx];
}

__device__ void moveToGlobal(volatile int *upperbuff, volatile int *dev_table, int tileAddress, int thread, int tileX, int tileY, int rowsize, int paddX, int segLengthX, int segLengthY, int warpbatch){
/*
	int idx = thread + segLengthX + paddX;
	int pos = tileAddress + (rowsize-1) + paddX + thread;
	//If y dimension cannot be completely divided by tileY, this code causes errors.
	if (thread < tileX){
		for (int i=0; i<tileY; i++){
			dev_table[pos] = table[idx];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
*/
	int idx = thread % 32;
	int warpidx = thread / 32;
	int glbpos = tileAddress + (rowsize - 1) + paddX + warpidx * (rowsize - 1);
	int shrpos = segLengthX + paddX + warpidx * segLengthX;

	for (; warpidx < tileY; warpidx += warpbatch){
		for (int i = idx; i < tileX; i += 32){
			dev_table[glbpos + i] = table[shrpos + i];	
		}
		shrpos += (warpbatch * segLengthX);
		glbpos += (warpbatch * (rowsize - 1) );
	}

}

__device__ void moveToShareRec(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int tileY, int rowsize, int segLengthX, int segLengthY, int warpbatch){
	//This function is designed for the first and the last tiles, which can be treate as rectangular but not hyperlane.
	//Rectangular tile does not have bank conflict issue.
	
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			table[idx] = dev_table[pos];
			pos += rowsize;
			idx += segLengthX;
		}	
	}
}


__device__ void moveToGlobalRec(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int tileY, int rowsize, int segLengthX, int segLengthY, int paddX){
	int pos = tileAddress + rowsize + paddX + thread;
	int idx = segLengthX + paddX + thread;
	if (thread < tileX){
		for (int i=0; i<tileY; i++){
			dev_table[pos] = table[idx];
			pos += rowsize;
			idx += segLengthX;
		}
	}	
}


__device__ void flagReadGlobal(int curBatch, volatile int *dev_lock, int thread, int idx, int YoverX, int xseg){
	if (thread == 0){
		int limit = min(idx+YoverX, xseg);
/*
		printf("curBatch: %d, tile: %d, limit: %d, dev_lock[curBatch]: %d\n", curBatch, idx, limit, dev_lock[curBatch]);
*/
	 	while(dev_lock[curBatch] < limit){
		}
/*		printf("curBatch: %d, tile: %d, is permit to proceed, dev_lock[curBatch]: %d\n", curBatch, idx, dev_lock[curBatch]);
*/
	}
	__syncthreads();
}

__device__ void flagWriteGlobal(int curBatch, volatile int *dev_lock, int thread){
	if (thread == 0){
		dev_lock[curBatch+1] += 1;
	}
	__syncthreads();
}

__global__ void GPU(volatile int *dev_table, int *dev_arr1, int *dev_arr2, volatile int *dev_lock, int curBatch, int curStartAddress, int rowtiles, int resX, int paddX, int paddY, int rowStartOffset, int rowsize, int colsize, int xseg, int yseg, int YoverX, int n1, int n2, int warpbatch){ 
	//We assume row size n2 is the multiple of 32 and can be completely divided by tileX.
	//on K40, tile size is max to 48K, which is 128*96; on pascal and volta, tile size is max to 64K which is 128*128
	//This code, length of x axis cannot be larger than y axis for each tile.
	//For each row, the first tile and the last tile are computed separately from the other tiles.
	//No padding added, thus the first tile of each row and the first row requires statement check to set dependency to 0 for the edge elements.
	//size of the shared memory is determined by the GPU architecture.
	
#ifdef DEBUG
	if (threadIdx.x == 0){
		printf("This is curBatch: %d, curStartAddress: %d\n", curBatch, curStartAddress);
	}
	__syncthreads();
#endif

	int warpsize = 32;
	int tileX = 64;
	int tileY = 128;
	volatile __shared__ int warplock[4];
	volatile __shared__ int	table[(tileY+1) * warpsize]; 
	__shared__ int arr1_seg[tileX * 2];
	__shared__ int arr2_seg[tileY];
	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int warp = thread / warpsize;
	int idx = thread % warpsize;
	int tileStartOffset, lvlStartAddress; 
	int glbStartX;
	int segLengthX = tileX + paddX;
	int segLengthY = tileY + paddY;
	volatile __shared__ int upperbuf[5 * tileX];
	int arrX = 0;
	int arrY = curBatch * tileY;
	int idxx, idxy, idx;
	int tile = 1;
	int prevx, prevy, prevxy, curr, iniUpp;
	
//***********************************************************************************************************************************
	//processing the first tile of the row
	glbStartX = curStartAddress;
	flagReadGlobal(curBatch, dev_lock, thread, tile, YoverX, xseg);
	
	int highY = tileY;
	int piece = tileY / tileX;

	for (int p = 0; p < piece; p++){
#ifdef DEBUG1
#ifdef ALL
		if (thread == 32 && curBatch == row ){	
#endif
#ifndef ALL
		if (thread == 32){
#endif
			printf("Before move data share memory. curBatch: %d, tile: %d, p: %d, xseg: %d, glbStartX: %d, segLengthX: %d, segLengthY: %d\n", curBatch, tile, p, xseg, glbStartX, segLengthX, segLengthY);
			//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[highY *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
		__syncthreads();
#endif	
	
		moveToShareRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, warpbatch);				
		__syncthreads();
//		__threadfence_system();

#ifdef DEBUG1
#ifdef ALL	
		if (thread == 32 && curBatch == row){	
#endif
#ifndef ALL
		if (thread == 32){
#endif	
			printf("Before computation, share memory. curBatch: %d, tile: %d, p: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, p, xseg, glbStartX);
			//for (int i=0; i<segLengthY; i++){

			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[highY *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
		__syncthreads();
#endif

		//first tile is irregular, concurrency is changed from 1 to hightY
		//the x length and y length of the first tile and the last tile are equal.
		tileStartOffset = paddY * segLengthX + paddX;
		//length Y > length X, diagonal first element starts from Y axis instead of X axis for calculating the address.
		int concurrency;
		for (int i=0; i<highY; i++){
			lvlStartAddress = tileStartOffset + i * segLengthX;
			concurrency = min(tileX, i);
			if(thread <= concurrency){
				idx = lvlStartAddress - segLengthX * thread + thread;
				idxx = arrX + thread;
				idxy = arrY + i - thread;
				table[idx] = max(table[idx-1], table[idx-segLengthX]);
				if(dev_arr1[idxx] == dev_arr2[idxy]){
					table[idx] = table[idx-segLengthX-1] + 1;	
				}	
	//			printf("curBatch: %d, tile: %d, thread: %d, idx: %d, idxx: %d, x[idxx]: %d, idxy: %d, y[idxy]: %d, table[idx]: %d\n", curBatch, tile, thread, idx, idxx, dev_arr1[idxx], idxy, dev_arr2[idxy], table[idx]);						
			}
			__syncthreads();
		}

#ifdef DEBUG1
#ifdef ALL	
		if (thread == 32 && curBatch==row){	
#endif
#ifndef ALL
		if (thread == 32){
#endif
			printf("After computation, in shared memory\n");
			//for (int i=0; i<segLengthY; i++){
				{	
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[highY *segLengthX+j]);
					}
					printf("\n");
					//for (int j = 0; j<segLengthX; j++){
					//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
					//}
					//printf("\n");
				}
			printf("\n");
		}
		__syncthreads();
#endif

		moveToGlobalRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, paddX);				
//		__threadfence_system();
//		__syncthreads();


#ifdef DEBUG1
#ifdef ALL
		if (thread == 32 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 32){
#endif
			printf("After computation in global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + highY *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
//			}
			printf("\n");
		}
		__syncthreads();
#endif
		arrX += tileX;		
		glbStartX += tileX;
		highY -= tileX;
//		__syncthreads();
	}
	//update the tile beginning pos for the next tile.
//	glbStartX += (tileY + 1);
	glbStartX += 1;
	arrX = tileY;
	flagWriteGlobal(curBatch, dev_lock, thread);
//***********************************************************************************************************************************
	//hyperlane tiles
	arr2_seg[thread] = dev_arr2[curBatch * tileY + thread];
	moveToSharePrev(&table[0], dev_table, glbStartX, warp, thread, idx, tileX, tileY, rowsize, segLengthX, segLengthY, warpbatch, warpsize);
	__syncthreads();

	prevx = table[(thread+1) * warpsize + paddX - 1];
	prevy = table[thread * warpsize + paddX - 2];
	prevxy = table[thread * warpsize + paddX - 3];

	for (tile = 2; tile < xseg; tile++){
		if (thread < tileX * 2)
			arr1_seg[thread] = dev_arr1[arrx - tileX + thread];
		flagReadGlobal(curBatch, dev_lock, thread, tile, YoverX, xseg);

		if (warp == 0)
			moveToShareUpper(&upperbuff[0], dev_table, glbStartX, thread, tileX, rowsize);
		__syncthreads();
		//inner loop for executing multiple 32*32 tiles in a row.
		for (int i=0; i< tileX; i+=warpsize){
			flagReadBlock(warp, &warplock[0], thread);
			for (int j=0; j<warpsize; j++){
				iniUpp = upperbuff[thread];
				
				if (thread == 0)
					prevy = iniUpp;
				curr = max(prevx, prevy);
				idxx = tileX + i - thread + j;
				idxy = thread;
				if (arr1_seg[idxx] == arr2_seg[idxy])
					curr = prevxy + 1;
				
				iniUpp = __shfl_down_sync(thread, iniUpp, 1, warpsize);
				prevx = curr;
				prevxy = prevy;
				prevy = __shfl_up_sync(thread, curr, 1, warpsize);
				
				if (thread == 31)
					upperbuf[(warp+1)*warpsize + j] = curr;   
			}
			flagWriteBlock();
		}
/*
#ifdef DEBUG2
#ifdef ALL
	if (thread == 0 && curBatch == row && tile<=3){	
#endif
#ifndef ALL
	if (thread == 0){
#endif
		printf("After computation, in shared memory.\n");
		//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-tileX) * segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
		printf("\n");
	}
	__syncthreads();
#endif
*/
		//need modification, only copy the new updated elements back to the global memory. Also modify moveToGlobalRec
		moveToGlobal(&upperbuff[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, paddX, segLengthX, segLengthY, warpbatch);
//		moveToGlobal(&table[paddX], dev_table, glbStartX + paddX, thread, tileX, rowsize, segLengthX, segLengthY);
		
//		__threadfence_system();
//		__syncthreads();

#ifdef DEBUG2
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation, global memory. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j - 1]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-tileX) *rowsize+j - (segLengthY-tileX-1)]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j - (segLengthY-1)]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j - i * (segLengthY-1)]);
				//}
				//printf("\n");
		//	}
			printf("\n");
		}
//		__syncthreads();
#endif

		//update the tile beginning pos for the next tile.
		glbStartX += tileX;
		arrX+=tileX;
		flagWriteGlobal(curBatch, dev_lock, thread);
	}

//************************************************************************************************************************************
	//the last tile, which is a half of the rectangular
	flagReadGlobal(curBatch, dev_lock, thread, xseg, YoverX, xseg);
	glbStartX = curStartAddress + rowsize - tileY - paddX;
	
	piece = tileY / tileX;
	highY = tileX;
	
	for (int p=0; p<piece; p++){
#ifdef DEBUG3
#ifdef ALL	
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif
			printf("Before computation global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY - 1) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY ) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY + 1) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i* (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
			//}
			printf("\n");
		}
		__syncthreads();
#endif	
		moveToShareRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, warpbatch);				
		__syncthreads();
//		__threadfence_block();

#ifdef DEBUG3
#ifdef ALL
			if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
			if (thread == 0){
#endif
				printf("last tile share memory before computation, glbStartX: %d, p: %d, rowsize: %d, segLengthY: %d\n", glbStartX, p, rowsize, segLengthY);	
			//	for (int i=0; i<segLengthY; i++){
				{	
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-highY - 1) * segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY + 1) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-1) *segLengthX+j]);
					}
					printf("\n");
					//for (int j = 0; j<segLengthX; j++){
					//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
					//}
					//printf("\n");
				}
				printf("\n");
			}
			__syncthreads();
#endif

		//last tile is irregular, concurrency is changed from hightY-1 to 1
		//the x length and y length of the first tile and the last tile are equal.
		int concurrency;
		tileStartOffset = segLengthX * (segLengthY - highY);
		for (int i=highY; i>0; i--){
			lvlStartAddress = tileStartOffset + segLengthX - 1;
			concurrency = min(tileX, i);	
			if(thread < concurrency){
				idx = lvlStartAddress + segLengthX * thread - thread;
				idxx = (n1 - tileY + highY - 1) - thread;
				//idxy = (n2 - i) + thread;
				idxy = (arrY + tileY - i) + thread;
				table[idx] = max(table[idx-1], table[idx-segLengthX]);
				if(dev_arr1[idxx] == dev_arr2[idxy]){
					table[idx] = table[idx-segLengthX-1] + 1;	
				}							
			}
			tileStartOffset += segLengthX;
			__syncthreads();
		}
	
		moveToGlobalRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, paddX);				

#ifdef DEBUG3
#ifdef ALL	
		if (thread == 0 && curBatch == row){	
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			__syncthreads();
			printf("After computation, in shared memory\n");
			//for (int i=0; i<segLengthY; i++){
				{	
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-highY - 1) * segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY + 1) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-1) *segLengthX+j]);
					}
					printf("\n");
					//for (int j = 0; j<segLengthX; j++){
					//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
					//}
					//printf("\n");
				}
			printf("\n");
		}
		__syncthreads();
#endif


#ifdef DEBUG3
#ifdef ALL
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation. global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + 2 * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
			//}
			printf("\n");
		}
		__syncthreads();
#endif
		glbStartX += tileX;
		highY += tileX;
	}
	
	flagWriteGlobal(curBatch, dev_lock, thread);
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

int LCS(int n1, int n2, int *arr1, int *arr2, int paddX, int paddY, int *table){
	hipSetDevice(0);	
	int lcslength;

	//tileY must be larger than tileX
	int tileX = 64;
	int tileY = 128;
	int rowsize = paddX + n2;
	int colsize = paddY + n1;

	int *dev_arr1, *dev_arr2;
	volatile int *dev_table, *dev_lock;
	int *lock;
	size_t freeMem, totalMem;
	
	
	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_table, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMalloc(&dev_arr1, n1*sizeof(int));
	hipMalloc(&dev_arr2, n2*sizeof(int));

	hipMemset((void*)dev_table, 0, tablesize * sizeof(int));
	hipMemcpy(dev_arr1, arr1, n1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr2, arr2, n2*sizeof(int), hipMemcpyHostToDevice);

//	int threadPerBlock = max(tileY + 32, tileX + 32);
	int threadPerBlock = 1024;
	int blockPerGrid = 1;
	int numStream = 28;
	int warpbatch = threadPerBlock / 32;

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	//For hyperlane tiles, if tileX!=tileY, the X length of the first tile and the last tile are equal to tileY.
//	int xseg = (n1+tileX-1) / tileX;
	int xseg = ((n1-tileY) + tileX - 1) / tileX + 2;
	int yseg = (n2+tileY-1) / tileY;

	lock = new int[yseg+1];
	lock[0] = xseg+1;
	for (int i=1; i<yseg+1; i++)
		lock[i] = 0;
	hipMalloc(&dev_lock, (yseg+1) * sizeof(int));	
//	hipMemset((void*)dev_lock, 0, (yseg + 1) * sizeof(int));
//	hipMemset((void*)dev_lock, xseg+1, sizeof(int));
	hipMemcpy((void*)dev_lock, lock, (yseg+1)*sizeof(int), hipMemcpyHostToDevice);
	hipStream_t stream[numStream];
	for (int s=0; s<numStream; s++)
		hipStreamCreate(&stream[s]);
	
	//instead of calling kernels along anti-diagonals, we now schedule kernels for each row.
	//We assume X axis is completly divided by tileX
	for(int curBatch = 0; curBatch < yseg; curBatch++){
		int curSMStream = curBatch % numStream;
		//int resY = n1 - curBatch * tileY;
		int resX = (n2 - tileY) % tileX;
		int curStartAddress = curBatch * tileY * rowsize;
		int rowStartOffset = paddY * rowsize + paddX;
		int rowtiles = xseg + 1;
//		cout << endl << "curBatch: " << curBatch << ", yseg: " << yseg << endl;	
		GPU<<<blockPerGrid, threadPerBlock, 0, stream[curSMStream]>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2, warpbatch);			
//		GPU<<<blockPerGrid, threadPerBlock>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
		checkGPUError( hipGetLastError() );
//		hipDeviceSynchronize();
	}
	hipMemcpy(&lcslength, (void*)&dev_table[tablesize-1], sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(table, (void*)dev_table, tablesize*sizeof(int), hipMemcpyDeviceToHost);

#ifdef DEBUG
	//display table
	cout << "grid size: " << blockPerGrid << ", block size: " << threadPerBlock << ", full table: " << endl;
	for (int i=0; i<colsize; i++){
		for (int j=0; j<rowsize; j++){
			cout << table[i * rowsize + j] << " ";
		}
		cout << endl;
	}
	
#endif
	for (int s=0; s<numStream; s++)
		hipStreamDestroy(stream[s]);
	
	hipFree(dev_arr1);
	hipFree(dev_arr2);
	hipFree((void*)dev_table);
	hipFree((void*)dev_lock);
	delete[] lock;

	return lcslength;
}

