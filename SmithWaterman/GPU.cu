
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

//#define debug
typedef unsigned long long int UINT;

using namespace std;

__device__ int s(int a, int b){
	return a==b?3:-3;
}

__global__ void GPU(int *dev_table, int *dev_arr1, int *dev_arr2, int startIdx, int curjobs, const int rowsize, int startx, int starty){
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (thread < curjobs){
		int idx = startIdx + (thread * rowsize - thread);
		int x = startx - thread;
		int y = starty + thread;
		
		dev_table[idx] = max(dev_table[idx-1]-2,
				max(dev_table[idx-rowsize]-2,
				max(dev_table[idx-rowsize-1] + s(dev_arr1[x], dev_arr2[y]), 0)));
	}
//	__threadfence();
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

int SW(int n1, int n2, int *arr1, int *arr2){
	int last;
	int paddsize = 1;
	int rowsize = paddsize + n2;
	int colsize = paddsize + n1;

	int *dev_table, *dev_arr1, *dev_arr2;

	int *table;

	table = new int[colsize * rowsize];

	size_t freeMem, totalMem;

	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_table, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMalloc(&dev_arr1, n1*sizeof(int));
	hipMalloc(&dev_arr2, n2*sizeof(int));

	hipMemset(dev_table, 0, tablesize * sizeof(int));
	hipMemcpy(dev_arr1, arr1, n1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr2, arr2, n2*sizeof(int), hipMemcpyHostToDevice);

	int maxthreads = min(n1, n2);
	int maxlevel = n1 + n2 - 1;	
	int curlevel = 1;
	int curjobs = 1;
	int startx, starty;
	int threadPerBlock = 32, blockPerGrid;

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	//suppose n2 is the row size and the longer array
	while(curlevel <= maxlevel){
//		cout << "level: " << curlevel << endl;
		int startIdx;
		if (curlevel <= n2){
			startIdx = curlevel - 1;
			curjobs = curlevel;
			startx = startIdx;
			starty = 0;
		}
		else{
			startIdx = n2 - 1 + rowsize * (curlevel - n2);
			curjobs = 2 * n2 - curlevel;
			startx = n2 - 1;
			starty = curlevel - n2;
		}

		int numthreads = (curjobs + 31) / 32;
		numthreads *= 32;
	
		blockPerGrid = (numthreads + threadPerBlock - 1) / threadPerBlock;

		GPU<<<blockPerGrid, threadPerBlock>>>(&dev_table[paddsize*rowsize+paddsize], dev_arr1, dev_arr2, startIdx, curjobs, rowsize, startx, starty);		
		
		hipDeviceSynchronize();		

		curlevel++;
	}

	hipMemcpy(&last, &dev_table[tablesize-1], sizeof(int), hipMemcpyDeviceToHost);
#ifdef debug
	cudaMemcpy(table, dev_table, (n1+paddsize)*rowsize*sizeof(int), cudaMemcpyDeviceToHost);
	//display table
	cout << "full table: " << endl;
	for (int i=0; i<n1+paddsize; i++){
		for (int j=0; j<n2+paddsize; j++){
			cout << table[i * rowsize + j] << " ";
		}
		cout << endl;
	}
#endif	

	hipFree(dev_arr1);
	hipFree(dev_arr2);
	hipFree(dev_table);

	delete[] table;

	return last;
}

