
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

//#define ALL
//#define DEBUG
//#define DEBUG1
//#define DEBUG2
//#define DEBUG3

using namespace std;
__device__ int row = 0;

__device__ void moveToShare(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int tileY, int rowsize, int segLengthX, int segLengthY, int warpbatch){
	//potential bank conflict for accessing the data of each anti-diagonal
	//to avoid bank conflict, have to adjust the memory layout by introducing dummy elements.
	//padding elements can be used as the dummy elements, but the number of padding of each dimension has to be an odd number.
/*
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			table[idx] = dev_table[pos];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
*/
	int idx = thread % 32;
	int warpidx = thread / 32;
	int glbpos = tileAddress + warpidx * (rowsize - 1);
	int shrpos = warpidx * segLengthX;
//	if (thread < segLengthX)
//		table[thread] = dev_table[tileAddress + thread];
	for (; warpidx < segLengthY; warpidx+=warpbatch){
		for (int i = idx; i < segLengthX; i+= 32){
			if (i < segLengthX){
				table[shrpos + i] = dev_table[glbpos + i];
			}
		}
		shrpos += (warpbatch * segLengthX);
		glbpos += (warpbatch * (rowsize - 1) );	
	}

}

__device__ void moveToGlobal(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int tileY, int rowsize, int paddX1, int segLengthX, int segLengthY, int warpbatch){
/*
	int idx = thread + segLengthX + paddX;
	int pos = tileAddress + (rowsize-1) + paddX + thread;
	//If y dimension cannot be completely divided by tileY, this code causes errors.
	if (thread < tileX){
		for (int i=0; i<tileY; i++){
			dev_table[pos] = table[idx];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
*/
	int idx = thread % 32;
	int warpidx = thread / 32;
	int glbpos = tileAddress + (rowsize - 1) + paddX1 + warpidx * (rowsize - 1);
	int shrpos = segLengthX + paddX1 + warpidx * segLengthX;

	for (; warpidx < tileY; warpidx += warpbatch){
		for (int i = idx; i < tileX; i += 32){
			dev_table[glbpos + i] = table[shrpos + i];	
		}
		shrpos += (warpbatch * segLengthX);
		glbpos += (warpbatch * (rowsize - 1) );
	}

}

__device__ void moveToShareRec(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int tileY, int rowsize, int segLengthX, int segLengthY, int warpbatch){
	//This function is designed for the first and the last tiles, which can be treate as rectangular but not hyperlane.
	//Rectangular tile does not have bank conflict issue.
	
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			table[idx] = dev_table[pos];
			pos += rowsize;
			idx += segLengthX;
		}	
	}
}


__device__ void moveToGlobalRec(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int tileY, int rowsize, int segLengthX, int segLengthY, int paddX1){
	int pos = tileAddress + rowsize + paddX1 + thread;
	int idx = segLengthX + paddX1 + thread;
	if (thread < tileX){
		for (int i=0; i<tileY; i++){
			dev_table[pos] = table[idx];
			pos += rowsize;
			idx += segLengthX;
		}
	}	
}


__device__ void flagRead(int curBatch, volatile int *dev_lock, int thread, int idx, int YoverX, int xseg){
	if (thread == 0){
		int limit = min(idx+YoverX, xseg);
	 	while(dev_lock[curBatch] < limit){
		}
	}
	__syncthreads();
}

__device__ void flagWrite(int curBatch, volatile int *dev_lock, int thread){
	if (thread == 0){
		dev_lock[curBatch+1] += 1;
	}
	__syncthreads();
}

__global__ void GPU(volatile int *dev_table, volatile int *dev_lock, int curBatch, int curStartAddress, int rowtiles, int resX, int tileX, int tileY, int paddX1, int paddX2, int paddY, int rowStartOffset, int rowsize, int colsize, int xseg, int yseg, int YoverX, int n1, int n2, int warpbatch){ 
	//We assume row size n2 is the multiple of 32 and can be completely divided by tileX.
	//on K40, tile size is max to 48K, which is 128*96; on pascal and volta, tile size is max to 64K which is 128*128
	//This code, length of x axis cannot be larger than y axis for each tile.
	//For each row, the first tile and the last tile are computed separately from the other tiles.
	//No padding added, thus the first tile of each row and the first row requires statement check to set dependency to 0 for the edge elements.
	//size of the shared memory is determined by the GPU architecture.
	
#ifdef DEBUG
	if (threadIdx.x == 0){
		printf("This is curBatch: %d, curStartAddress: %d\n", curBatch, curStartAddress);
	}
	__syncthreads();
#endif

	volatile __shared__ int table[12288];

	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int tileStartOffset, lvlStartAddress; 
	int glbStartX;
	int segLengthX = tileX + paddX1 + paddX2;
	int segLengthY = tileY + paddY + paddY;
	int idx;
	int tile = 1;

//***********************************************************************************************************************************
	//processing the first tile of the row
	glbStartX = curStartAddress;
	flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
	
	int highY = tileY;
	int piece = tileY / tileX;

	for (int p = 0; p < piece; p++){
		moveToShareRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, warpbatch);				
		__syncthreads();

		//first tile is irregular, concurrency is changed from 1 to hightY
		//the x length and y length of the first tile and the last tile are equal.
		tileStartOffset = paddY * segLengthX + paddX1;
		//length Y > length X, diagonal first element starts from Y axis instead of X axis for calculating the address.
		int concurrency;
		for (int i=0; i<highY; i++){
			lvlStartAddress = tileStartOffset + i * segLengthX;
			concurrency = min(tileX, i);
			if(thread <= concurrency){
				idx = lvlStartAddress - segLengthX * thread + thread;
				table[idx] = table[idx-1] + table[idx-segLengthX] + table[idx] - table[idx-segLengthX-1];
			}
			__syncthreads();
		}

		moveToGlobalRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, paddX1);	
		glbStartX += tileX;
		highY -= tileX;
	}
	//update the tile beginning pos for the next tile.
	glbStartX += 1;
	flagWrite(curBatch, dev_lock, thread);
//***********************************************************************************************************************************
	//hyperlane tiles
	tileStartOffset = paddY * segLengthX + paddX1;
	for (tile = 2; tile < xseg; tile++){
		flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
		
		moveToShare(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, warpbatch);
		__syncthreads();
		
		lvlStartAddress = tileStartOffset;
		for (int i=0; i<tileX; i++){
//this is expensive especially when tileX is large. However, if we put if statement outside the loop, we face syncthreads issue.
//New feature warp level syncronize or thread group syncronize may solve the problem.
			if (thread < tileY){
				idx = lvlStartAddress + thread * segLengthX;
				table[idx] = table[idx-1] + table[idx-segLengthX] + table[idx] - table[idx-segLengthX-1];
				lvlStartAddress += 1;
			}
			__syncthreads();
		}
		//need modification, only copy the new updated elements back to the global memory. Also modify moveToGlobalRec
		moveToGlobal(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, paddX1, segLengthX, segLengthY, warpbatch);
//		moveToGlobal(&table[paddX], dev_table, glbStartX + paddX, thread, tileX, rowsize, segLengthX, segLengthY);

		//update the tile beginning pos for the next tile.
		glbStartX += tileX;
		flagWrite(curBatch, dev_lock, thread);
	}

//************************************************************************************************************************************
	//the last tile, which is a half of the rectangular
	flagRead(curBatch, dev_lock, thread, xseg, YoverX, xseg);
	glbStartX = curStartAddress + rowsize - paddX2 - tileY - paddX1;
	
	piece = tileY / tileX;
	highY = tileX;
	
	for (int p=0; p<piece; p++){
		moveToShareRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, warpbatch);				
		__syncthreads();

		//last tile is irregular, concurrency is changed from hightY-1 to 1
		//the x length and y length of the first tile and the last tile are equal.
		int concurrency;
		tileStartOffset = segLengthX * (segLengthY - highY);
		for (int i=highY; i>0; i--){
			lvlStartAddress = tileStartOffset + segLengthX - 1;
			concurrency = min(tileX, i);	
			if(thread < concurrency){
				idx = lvlStartAddress + segLengthX * thread - thread;
				table[idx] = table[idx-1] + table[idx-segLengthX] + table[idx] - table[idx-segLengthX-1];
			}
			tileStartOffset += segLengthX;
			__syncthreads();
		}
	
		moveToGlobalRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, paddX1);	
		glbStartX += tileX;
		highY += tileX;
	}
	
	flagWrite(curBatch, dev_lock, thread);
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void SOR(int n1, int n2, int *table, int tX, int tY){
	hipSetDevice(0);	
	hipDeviceProp_t gpuinfo;
	hipGetDeviceProperties(&gpuinfo, 0);
	int paddX1 = 1, paddX2 = 2;
	int paddY = 1;
	//tileY must be larger than tileX
	int tileX = tX;
	int tileY = tY;
	int rowsize = paddX1 + paddX2 + n1;
	int colsize = 2 * paddY + n2;

	volatile int *dev_table, *dev_lock;
	int *lock;
	size_t freeMem, totalMem;
	
	
	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
#ifdef DEBUG
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
#endif	
	hipError_t err = hipMalloc(&dev_table, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMemcpy((void*)dev_table, table, tablesize*sizeof(int), hipMemcpyHostToDevice);

//	int threadPerBlock = max(tileY + 32, tileX + 32);
	int threadPerBlock = 1024;
	int blockPerGrid = 1;
	int numStream = gpuinfo.multiProcessorCount;
	int warpbatch = threadPerBlock / 32;

	//For hyperlane tiles, if tileX!=tileY, the X length of the first tile and the last tile are equal to tileY.
//	int xseg = (n1+tileX-1) / tileX;
	int xseg = ((n1-tileY) + tileX - 1) / tileX + 2;
	int yseg = (n2+tileY-1) / tileY;

	lock = new int[yseg+1];
	lock[0] = xseg+1;
	for (int i=1; i<yseg+1; i++)
		lock[i] = 0;
	hipMalloc(&dev_lock, (yseg+1) * sizeof(int));	
	hipMemcpy((void*)dev_lock, lock, (yseg+1)*sizeof(int), hipMemcpyHostToDevice);
	hipStream_t stream[numStream];
	for (int s=0; s<numStream; s++)
		hipStreamCreate(&stream[s]);
	
	//instead of calling kernels along anti-diagonals, we now schedule kernels for each row.
	//We assume X axis is completly divided by tileX
	for(int curBatch = 0; curBatch < yseg; curBatch++){
		int curSMStream = curBatch % numStream;
		//int resY = n1 - curBatch * tileY;
		int resX = (n2 - tileY) % tileX;
		int curStartAddress = curBatch * tileY * rowsize;
		int rowStartOffset = (paddX1+paddX2) * rowsize + paddX1;
		int rowtiles = xseg + 1;
//		cout << endl << "curBatch: " << curBatch << ", yseg: " << yseg << endl;	
		GPU<<<blockPerGrid, threadPerBlock, 0, stream[curSMStream]>>>(dev_table, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY,  paddX1, paddX2, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2, warpbatch);			
//		GPU<<<blockPerGrid, threadPerBlock>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
		checkGPUError( hipGetLastError() );
//		cudaDeviceSynchronize();
	}
	hipDeviceSynchronize();	
//cudaMemcpy(table, (void*)dev_table, tablesize*sizeof(int), cudaMemcpyDeviceToHost);

#ifdef DEBUG
	//display table
	cout << "grid size: " << blockPerGrid << ", block size: " << threadPerBlock << ", full table: " << endl;
	for (int i=0; i<colsize; i++){
		for (int j=0; j<rowsize; j++){
			cout << table[i * rowsize + j] << " ";
		}
		cout << endl;
	}
	
#endif
	for (int s=0; s<numStream; s++)
		hipStreamDestroy(stream[s]);
	
	hipFree((void*)dev_table);
	hipFree((void*)dev_lock);
	delete[] lock;

}

