
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

//#define PRINT_FINAL_RESULT
typedef unsigned long long int UINT;

using namespace std;

__global__ void GPU(int *dev_arr1, int *dev_arr2, const int rowsize, 
			const int colsize, const int n1, const int threadsPerBlock, int padd){
	int offset = rowsize * blockIdx.x + padd;
	int idx = threadIdx.x + offset;
	while (idx < n1 + padd + offset){
		dev_arr2[idx] = (dev_arr1[idx-1] + dev_arr1[idx-rowsize] + dev_arr1[idx]
				+ dev_arr1[idx+1] + dev_arr1[idx+rowsize]) / 5;	
		idx += threadsPerBlock;
	}	
	__threadfence();	
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void SOR(int n1, int n2, int padd, int *arr1, int *arr2, int MAXTRIAL){
	int rowsize = n1 + 2 * padd;
	int colsize = n2 + 2 * padd;

	int *dev_arr1, *dev_arr2, *tmp;
	int tablesize = rowsize * colsize;
	
//	size_t freeMem, totalMem;
//	cudaMemGetInfo(&freeMem, &totalMem);
//	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
//	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_arr1, tablesize * sizeof(int));
	checkGPUError(err);
	err = hipMalloc(&dev_arr2, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMemcpy(dev_arr1, arr1, tablesize * sizeof(int), hipMemcpyHostToDevice);

	int threadsPerBlock = min(1024, n1);
	int blocksPerGrid = n2;

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	struct timeval tbegin, tend;
	gettimeofday(&tbegin, NULL);

	//suppose n1 is the row size and the longer array
	for (int t = 0; t < MAXTRIAL; t++){
		GPU<<<blocksPerGrid, threadsPerBlock>>>(&dev_arr1[padd * rowsize], &dev_arr2[padd * rowsize], rowsize, colsize, n1, threadsPerBlock, padd);		
		hipDeviceSynchronize();
		tmp = dev_arr1;
		dev_arr1 = dev_arr2;
		dev_arr2 = tmp;
	}

	gettimeofday(&tend, NULL);
	double s = (double)(tend.tv_sec - tbegin.tv_sec) + (double)(tend.tv_usec - tbegin.tv_usec) / 1000000.0;
	cout << "execution time: " << s << " second." << endl;

	hipMemcpy(arr1, dev_arr1, tablesize*sizeof(int), hipMemcpyDeviceToHost);
#ifdef PRINT_FINAL_RESULT
	//display table
	cout << "full table: " << endl;
	for (int i=0; i<colsize; i++){
		for (int j=0; j<rowsize; j++){
			cout << arr1[i * rowsize + j] << " ";
		}
		cout << endl;
	}
#endif
	cout << "The last element: " << arr1[n2*rowsize + n1] << endl;
	
	hipFree(dev_arr1);
	hipFree(dev_arr2);
}

