
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

//#define PRINT_FINAL_RESULT
typedef unsigned long long int UINT;

using namespace std;

__device__ void _5pt_SOR(int* dev_arr1, int* dev_arr2, int idx, int rowsize){
	dev_arr2[idx] = (dev_arr1[idx-1] + dev_arr1[idx-rowsize] + dev_arr1[idx] + dev_arr1[idx+1] + dev_arr1[idx+rowsize]) / 5;	
}

__device__ void _9pt_SQUARE_SOR(int* dev_arr1, int* dev_arr2, int idx, int rowsize){
	dev_arr2[idx] = (dev_arr1[idx - rowsize - 1] + dev_arr1[idx - rowsize] + dev_arr1[idx - rowsize + 1] + dev_arr1[idx-1] + dev_arr1[idx] + dev_arr1[idx + 1] + dev_arr1[idx + rowsize - 1] + dev_arr1[idx+rowsize] + dev_arr1[idx + rowsize + 1]) / 9;	
}

__device__ void _9pt_CROSS_SOR(int* dev_arr1, int* dev_arr2, int idx, int rowsize){
	dev_arr2[idx] = (dev_arr1[idx - rowsize - rowsize] + dev_arr1[idx - rowsize] + dev_arr1[idx] + dev_arr1[idx-1] + dev_arr1[idx - 2] + dev_arr1[idx + 1] + dev_arr1[idx + 2] + dev_arr1[idx + rowsize] + dev_arr1[idx + rowsize + rowsize]) / 9;	
}

__device__ void _25pt_SQUARE_SOR(int* dev_arr1, int* dev_arr2, int idx, int rowsize){
	int total = 0;
	for (int i = -2; i <= 2; i++){
		for (int j = -2; j <= 2; j++){
			total += dev_arr1[idx + i * rowsize + j];
		}
	}
	dev_arr2[idx] = total / 9;
}
	
__global__ void GPU(int *dev_arr1, int *dev_arr2, const int rowsize, 
			const int colsize, const int n1, const int threadsPerBlock, int padd){
	int offset = rowsize * blockIdx.x + padd;
	int idx = threadIdx.x + offset;
	while (idx < n1 + offset){
//		_5pt_SOR(dev_arr1, dev_arr2, idx, rowsize);
//		_9pt_SQUARE_SOR(dev_arr1, dev_arr2, idx, rowsize);
//		_9pt_CROSS_SOR(dev_arr1, dev_arr2, idx, rowsize);
		_25pt_SQUARE_SOR(dev_arr1, dev_arr2, idx, rowsize);

		idx += threadsPerBlock;
	}	
	__threadfence();	
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void SOR(int n1, int n2, int padd, int *arr1, int *arr2, int MAXTRIAL){
	int rowsize = n1 + 2 * padd;
	int colsize = n2 + 2 * padd;

	int *dev_arr1, *dev_arr2, *tmp;
	int tablesize = rowsize * colsize;
	
//	size_t freeMem, totalMem;
//	cudaMemGetInfo(&freeMem, &totalMem);
//	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
//	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_arr1, tablesize * sizeof(int));
	checkGPUError(err);
	err = hipMalloc(&dev_arr2, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMemcpy(dev_arr1, arr1, tablesize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr2, arr1, tablesize * sizeof(int), hipMemcpyHostToDevice);

	int threadsPerBlock = min(1024, n1);
	int blocksPerGrid = n2;

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	struct timeval tbegin, tend;
	gettimeofday(&tbegin, NULL);

	//suppose n1 is the row size and the longer array
	for (int t = 0; t < MAXTRIAL; t++){
		GPU<<<blocksPerGrid, threadsPerBlock>>>(&dev_arr1[padd * rowsize], &dev_arr2[padd * rowsize], rowsize, colsize, n1, threadsPerBlock, padd);		
		hipDeviceSynchronize();
		tmp = dev_arr1;
		dev_arr1 = dev_arr2;
		dev_arr2 = tmp;
	}

	gettimeofday(&tend, NULL);
	double s = (double)(tend.tv_sec - tbegin.tv_sec) + (double)(tend.tv_usec - tbegin.tv_usec) / 1000000.0;

	hipMemcpy(arr1, dev_arr1, tablesize*sizeof(int), hipMemcpyDeviceToHost);
#ifdef PRINT_FINAL_RESULT
	//display table
	cout << "result table: " << endl;
	for (int i=0; i<colsize; i++){
		for (int j=0; j<rowsize; j++){
			cout << arr1[i * rowsize + j] << " ";
		}
		cout << endl;
	}
#endif
	cout << "execution time: " << s << " second." << endl;
	
	hipFree(dev_arr1);
	hipFree(dev_arr2);
}

