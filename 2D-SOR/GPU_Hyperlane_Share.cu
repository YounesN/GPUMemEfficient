
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

//#define ALL
//#define DEBUG
//#define DEBUG1
//#define DEBUG2
//#define DEBUG3

using namespace std;
__device__ int row = 0;

__device__ void moveToShare(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int tileY, int rowsize, int segLengthX, int segLengthY, int warpbatch){
	//potential bank conflict for accessing the data of each anti-diagonal
	//to avoid bank conflict, have to adjust the memory layout by introducing dummy elements.
	//padding elements can be used as the dummy elements, but the number of padding of each dimension has to be an odd number.
/*
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			table[idx] = dev_table[pos];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
*/
	int idx = thread % 32;
	int warpidx = thread / 32;
	int glbpos = tileAddress + (rowsize - 1) + warpidx * (rowsize - 1) + idx;
	int shrpos = segLengthX + warpidx * segLengthX + idx;
	if (thread < segLengthX)
		table[thread] = dev_table[tileAddress + thread];
	for (; warpidx < tileY; warpidx+=warpbatch){
		table[shrpos] = dev_table[glbpos];
		shrpos += (warpbatch * segLengthX);
		glbpos += (warpbatch * (rowsize - 1) );
	}

}

__device__ void moveToGlobal(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int tileY, int rowsize, int paddX, int segLengthX, int segLengthY, int warpbatch){
/*
	int idx = thread + segLengthX + paddX;
	int pos = tileAddress + (rowsize-1) + paddX + thread;
	//If y dimension cannot be completely divided by tileY, this code causes errors.
	if (thread < tileX){
		for (int i=0; i<tileY; i++){
			dev_table[pos] = table[idx];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
*/
	int idx = thread % 32;
	int warpidx = thread / 32;
	int glbpos = tileAddress + (rowsize - 1) + paddX + warpidx * (rowsize - 1);
	int shrpos = segLengthX + paddX + warpidx * segLengthX;

	for (; warpidx < tileY; warpidx += warpbatch){
		for (int i = idx; i < tileX; i += 32){
			dev_table[glbpos + i] = table[shrpos + i];	
		}
		shrpos += (warpbatch * segLengthX);
		glbpos += (warpbatch * (rowsize - 1) );
	}

}

__device__ void moveToShareRec(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int tileY, int rowsize, int segLengthX, int segLengthY, int warpbatch){
	//This function is designed for the first and the last tiles, which can be treate as rectangular but not hyperlane.
	//Rectangular tile does not have bank conflict issue.
	
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			table[idx] = dev_table[pos];
			pos += rowsize;
			idx += segLengthX;
		}	
	}
}


__device__ void moveToGlobalRec(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int tileY, int rowsize, int segLengthX, int segLengthY, int paddsize){
	int pos = tileAddress + rowsize + paddsize + thread;
	int idx = segLengthX + paddsize + thread;
	if (thread < tileX){
		for (int i=0; i<tileY; i++){
			dev_table[pos] = table[idx];
			pos += rowsize;
			idx += segLengthX;
		}
	}	
}


__device__ void flagRead(int curBatch, volatile int *dev_lock, int thread, int idx, int YoverX, int xseg){
	if (thread == 0){
		int limit = min(idx+YoverX, xseg);
/*
		printf("curBatch: %d, tile: %d, limit: %d, dev_lock[curBatch]: %d\n", curBatch, idx, limit, dev_lock[curBatch]);
*/
	 	while(dev_lock[curBatch] < limit){
		}
/*		printf("curBatch: %d, tile: %d, is permit to proceed, dev_lock[curBatch]: %d\n", curBatch, idx, dev_lock[curBatch]);
*/
	}
	__syncthreads();
}

__device__ void flagWrite(int curBatch, volatile int *dev_lock, int thread){
	if (thread == 0){
		dev_lock[curBatch+1] += 1;
	}
	__syncthreads();
}

__global__ void GPU(volatile int *dev_table, volatile int *dev_lock, int curBatch, int curStartAddress, int rowtiles, int resX, int tileX, int tileY, int paddsize, int rowStartOffset, int rowsize, int colsize, int xseg, int yseg, int YoverX, int n1, int n2, int warpbatch){ 
	//We assume row size n2 is the multiple of 32 and can be completely divided by tileX.
	//on K40, tile size is max to 48K, which is 128*96; on pascal and volta, tile size is max to 64K which is 128*128
	//This code, length of x axis cannot be larger than y axis for each tile.
	//For each row, the first tile and the last tile are computed separately from the other tiles.
	//No padding added, thus the first tile of each row and the first row requires statement check to set dependency to 0 for the edge elements.
	//size of the shared memory is determined by the GPU architecture.
	
#ifdef DEBUG
	if (threadIdx.x == 0){
		printf("This is curBatch: %d, curStartAddress: %d\n", curBatch, curStartAddress);
	}
	__syncthreads();
#endif

	volatile __shared__ int table[12288];

	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int tileStartOffset, lvlStartAddress; 
	int glbStartX;
	int segLengthX = tileX + 2 * paddsize;
	int segLengthY = tileY + 2 * paddsize;
	int idx;
	int tile = 1;

//***********************************************************************************************************************************
	//processing the first tile of the row
	glbStartX = curStartAddress;
	flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
	
	int highY = tileY;
	int piece = tileY / tileX;

	for (int p = 0; p < piece; p++){
#ifdef DEBUG1
#ifdef ALL
		if (thread == 32 && curBatch == row ){	
#endif
#ifndef ALL
		if (thread == 32){
#endif
			printf("Before move data share memory. curBatch: %d, tile: %d, p: %d, xseg: %d, glbStartX: %d, segLengthX: %d, segLengthY: %d\n", curBatch, tile, p, xseg, glbStartX, segLengthX, segLengthY);
			//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[highY *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
		__syncthreads();
#endif	
	
		moveToShareRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, warpbatch);				
		__syncthreads();
//		__threadfence_system();

#ifdef DEBUG1
#ifdef ALL	
		if (thread == 32 && curBatch == row){	
#endif
#ifndef ALL
		if (thread == 32){
#endif	
			printf("Before computation, share memory. curBatch: %d, tile: %d, p: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, p, xseg, glbStartX);
			//for (int i=0; i<segLengthY; i++){

			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[highY *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
		__syncthreads();
#endif

		//first tile is irregular, concurrency is changed from 1 to hightY
		//the x length and y length of the first tile and the last tile are equal.
		tileStartOffset = paddsize * segLengthX + paddsize;
		//length Y > length X, diagonal first element starts from Y axis instead of X axis for calculating the address.
		int concurrency;
		for (int i=0; i<highY; i++){
			lvlStartAddress = tileStartOffset + i * segLengthX;
			concurrency = min(tileX, i);
			if(thread <= concurrency){
				idx = lvlStartAddress - segLengthX * thread + thread;
				table[idx] = (table[idx-1] + table[idx-segLengthX] + table[idx]
						+ table[idx+1] + table[idx+segLengthX]) / 5;
			}
			__syncthreads();
		}

#ifdef DEBUG1
#ifdef ALL	
		if (thread == 32 && curBatch==row){	
#endif
#ifndef ALL
		if (thread == 32){
#endif
			printf("After computation, in shared memory\n");
			//for (int i=0; i<segLengthY; i++){
				{	
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[highY *segLengthX+j]);
					}
					printf("\n");
					//for (int j = 0; j<segLengthX; j++){
					//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
					//}
					//printf("\n");
				}
			printf("\n");
		}
		__syncthreads();
#endif

		moveToGlobalRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, paddsize);				
//		__threadfence_system();
//		__syncthreads();


#ifdef DEBUG1
#ifdef ALL
		if (thread == 32 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 32){
#endif
			printf("After computation in global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + highY *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
//			}
			printf("\n");
		}
		__syncthreads();
#endif
		glbStartX += tileX;
		highY -= tileX;
	}
	//update the tile beginning pos for the next tile.
	glbStartX += 1;
	flagWrite(curBatch, dev_lock, thread);
//***********************************************************************************************************************************
	//hyperlane tiles
	tileStartOffset = paddsize * segLengthX + paddsize;
	for (tile = 2; tile < xseg; tile++){
		flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
#ifdef DEBUG2
#ifdef ALL		
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("Before computation global memory. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j - 1]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j - (segLengthY-1)]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j - i * (segLengthY-1)]);
				//}
				//printf("\n");
//			}
			printf("\n");
		}
		__syncthreads();
#endif	
#ifdef DEBUG2
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3 ){	
#endif
#ifndef ALL
		if (thread == 0){
#endif
			printf("Before move data to share memory. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
		__syncthreads();
#endif	
		moveToShare(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, warpbatch);
		__syncthreads();
	//	__threadfence_system();
#ifdef DEBUG2
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3 ){	
#endif
#ifndef ALL
		if (thread == 0){
#endif
			printf("Before computation share. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-tileX) *segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
		__syncthreads();
#endif		
		lvlStartAddress = tileStartOffset;
		for (int i=0; i<tileX; i++){
//this is expensive especially when tileX is large. However, if we put if statement outside the loop, we face syncthreads issue.
//New feature warp level syncronize or thread group syncronize may solve the problem.
			if (thread < tileY){
				idx = lvlStartAddress + thread * segLengthX;
				table[idx] = (table[idx-1] + table[idx-segLengthX] + table[idx]
						+ table[idx+1] + table[idx+segLengthX]) / 5;
				lvlStartAddress += 1;
			}
			__syncthreads();
		}
#ifdef DEBUG2
#ifdef ALL
	if (thread == 0 && curBatch == row && tile<=3){	
#endif
#ifndef ALL
	if (thread == 0){
#endif
		printf("After computation, in shared memory.\n");
		//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-tileX) * segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
		printf("\n");
	}
	__syncthreads();
#endif
		//need modification, only copy the new updated elements back to the global memory. Also modify moveToGlobalRec
		moveToGlobal(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, paddsize, segLengthX, segLengthY, warpbatch);
//		moveToGlobal(&table[paddX], dev_table, glbStartX + paddX, thread, tileX, rowsize, segLengthX, segLengthY);
		
//		__threadfence_system();
//		__syncthreads();

#ifdef DEBUG2
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation, global memory. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j - 1]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-tileX) *rowsize+j - (segLengthY-tileX-1)]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j - (segLengthY-1)]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j - i * (segLengthY-1)]);
				//}
				//printf("\n");
		//	}
			printf("\n");
		}
//		__syncthreads();
#endif

		//update the tile beginning pos for the next tile.
		glbStartX += tileX;
		flagWrite(curBatch, dev_lock, thread);
	}

//************************************************************************************************************************************
	//the last tile, which is a half of the rectangular
	flagRead(curBatch, dev_lock, thread, xseg, YoverX, xseg);
	glbStartX = curStartAddress + rowsize - paddsize - tileY - paddsize;
	
	piece = tileY / tileX;
	highY = tileX;
	
	for (int p=0; p<piece; p++){
#ifdef DEBUG3
#ifdef ALL	
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif
			printf("Before computation global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY - 1) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY ) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY + 1) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i* (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
			//}
			printf("\n");
		}
		__syncthreads();
#endif	
		moveToShareRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, warpbatch);				
		__syncthreads();
//		__threadfence_block();

#ifdef DEBUG3
#ifdef ALL
			if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
			if (thread == 0){
#endif
				printf("last tile share memory before computation, glbStartX: %d, p: %d, rowsize: %d, segLengthY: %d\n", glbStartX, p, rowsize, segLengthY);	
			//	for (int i=0; i<segLengthY; i++){
				{	
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-highY - 1) * segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY + 1) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-1) *segLengthX+j]);
					}
					printf("\n");
					//for (int j = 0; j<segLengthX; j++){
					//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
					//}
					//printf("\n");
				}
				printf("\n");
			}
			__syncthreads();
#endif

		//last tile is irregular, concurrency is changed from hightY-1 to 1
		//the x length and y length of the first tile and the last tile are equal.
		int concurrency;
		tileStartOffset = segLengthX * (segLengthY - highY);
		for (int i=highY; i>0; i--){
			lvlStartAddress = tileStartOffset + segLengthX - 1;
			concurrency = min(tileX, i);	
			if(thread < concurrency){
				idx = lvlStartAddress + segLengthX * thread - thread;
				table[idx] = (table[idx-1] + table[idx-segLengthX] + table[idx]
						+ table[idx+1] + table[idx+segLengthX]) / 5;
			}
			tileStartOffset += segLengthX;
			__syncthreads();
		}
	
		moveToGlobalRec(&table[0], dev_table, glbStartX, thread, tileX, tileY, rowsize, segLengthX, segLengthY, paddsize);				

#ifdef DEBUG3
#ifdef ALL	
		if (thread == 0 && curBatch == row){	
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			__syncthreads();
			printf("After computation, in shared memory\n");
			//for (int i=0; i<segLengthY; i++){
				{	
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-highY - 1) * segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY + 1) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-1) *segLengthX+j]);
					}
					printf("\n");
					//for (int j = 0; j<segLengthX; j++){
					//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
					//}
					//printf("\n");
				}
			printf("\n");
		}
		__syncthreads();
#endif


#ifdef DEBUG3
#ifdef ALL
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation. global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + 2 * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
			//}
			printf("\n");
		}
		__syncthreads();
#endif
		glbStartX += tileX;
		highY += tileX;
	}
	
	flagWrite(curBatch, dev_lock, thread);
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void SOR(int n1, int n2, int *table){
	hipSetDevice(0);	
	int paddsize = 1;
	//tileY must be larger than tileX
	int tileX = 64;
	int tileY = 128;
	int rowsize = 2 * paddsize + n1;
	int colsize = 2 * paddsize + n2;

	volatile int *dev_table, *dev_lock;
	int *lock;
	size_t freeMem, totalMem;
	
	
	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_table, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMemcpy((void*)dev_table, table, tablesize*sizeof(int), hipMemcpyHostToDevice);

//	int threadPerBlock = max(tileY + 32, tileX + 32);
	int threadPerBlock = 1024;
	int blockPerGrid = 1;
	int numStream = 28;
	int warpbatch = threadPerBlock / 32;

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	//For hyperlane tiles, if tileX!=tileY, the X length of the first tile and the last tile are equal to tileY.
//	int xseg = (n1+tileX-1) / tileX;
	int xseg = ((n1-tileY) + tileX - 1) / tileX + 2;
	int yseg = (n2+tileY-1) / tileY;

	lock = new int[yseg+1];
	lock[0] = xseg+1;
	for (int i=1; i<yseg+1; i++)
		lock[i] = 0;
	hipMalloc(&dev_lock, (yseg+1) * sizeof(int));	
	hipMemcpy((void*)dev_lock, lock, (yseg+1)*sizeof(int), hipMemcpyHostToDevice);
	hipStream_t stream[numStream];
	for (int s=0; s<numStream; s++)
		hipStreamCreate(&stream[s]);
	
	//instead of calling kernels along anti-diagonals, we now schedule kernels for each row.
	//We assume X axis is completly divided by tileX
	for(int curBatch = 0; curBatch < yseg; curBatch++){
		int curSMStream = curBatch % numStream;
		//int resY = n1 - curBatch * tileY;
		int resX = (n2 - tileY) % tileX;
		int curStartAddress = curBatch * tileY * rowsize;
		int rowStartOffset = paddsize * rowsize + paddsize;
		int rowtiles = xseg + 1;
//		cout << endl << "curBatch: " << curBatch << ", yseg: " << yseg << endl;	
		GPU<<<blockPerGrid, threadPerBlock, 0, stream[curSMStream]>>>(dev_table, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY,  paddsize, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2, warpbatch);			
//		GPU<<<blockPerGrid, threadPerBlock>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
		checkGPUError( hipGetLastError() );
//		cudaDeviceSynchronize();
	}
	hipDeviceSynchronize();	
//cudaMemcpy(table, (void*)dev_table, tablesize*sizeof(int), cudaMemcpyDeviceToHost);

#ifdef DEBUG
	//display table
	cout << "grid size: " << blockPerGrid << ", block size: " << threadPerBlock << ", full table: " << endl;
	for (int i=0; i<colsize; i++){
		for (int j=0; j<rowsize; j++){
			cout << table[i * rowsize + j] << " ";
		}
		cout << endl;
	}
	
#endif
	for (int s=0; s<numStream; s++)
		hipStreamDestroy(stream[s]);
	
	hipFree((void*)dev_table);
	hipFree((void*)dev_lock);
	delete[] lock;

}

