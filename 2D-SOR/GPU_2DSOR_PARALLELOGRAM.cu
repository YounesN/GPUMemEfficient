
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

//#define ALL
//#define DEBUG
//#define DEBUG1
//#define DEBUG2
//#define DEBUG3

using namespace std;
__device__ int row = 0;

__device__ void moveMatrixToTile(volatile int* dev_arr, int* tile, int segLengthX, int tileX, int tileY, int dep_stride, int tileAddress, int rowsize, int warpbatch, int thread){
	int idx = thread % 32;
	int warpidx = thread / 32;
	int glbpos = tileAddress + warpidx * rowsize + idx;
	int shrpos = dep_stride * segLengthX + warpidx * segLengthX + dep_stride + idx;
//	if (thread < segLengthX)
//		table[thread] = dev_table[tileAddress + thread];
	for (; warpidx < tileY; warpidx += warpbatch){
		for (int i = idx; i < tileX; i += 32){
			tile[shrpos+i] = dev_arr[glbpos+i];
		}
		shrpos += (warpbatch * segLengthX);
		glbpos += (warpbatch * rowsize);
	}
}

//intra_dep array structure: tileT * dep_stride * tileY
__device__ void moveIntraDepToTile(int* intra_dep, int* tile, int tt, int tileY, int segLengthX, int dep_stride, int thread, int len){
	//at each tt, (stride+1) dependent data are required at x axis.
	//only the threads, which are within tileY are working here.
	//threadPerBlock has to be no less than tileY * dep_stride
	if (thread < len * dep_stride){
		int pos = tt * dep_stride * tileY + thread;
		int tilepos = dep_stride * segLengthX + thread/dep_stride * segLengthX + thread % dep_stride;
		tile[tilepos] = intra_dep[pos];
	}
}

__device__ void moveIntraDepToTileEdge(volatile int* dev_arr, int* tile, int stride, int rowsize, int segLengthX, int dep_stride, int thread, int tt, int padd, int n1, int len, int offset = 0){
	//copy out-of-range data to tile
	if (thread < len * dep_stride){
		int glbpos = padd * rowsize + (padd - dep_stride) + offset * (n1 + dep_stride) + thread/dep_stride * rowsize + thread % dep_stride;
		int tilepos = dep_stride * segLengthX + thread/dep_stride * segLengthX + thread % dep_stride + offset * (dep_stride + tt);
		tile[tilepos] = dev_arr[glbpos];
	}
}

__device__ void moveTileToIntraDep(int* intra_dep, int* tile, int tt, int tileX, int tileY, int segLengthX, int dep_stride, int thread, int isRegular, int len){
	if (thread < len * dep_stride){
		int pos = tt * dep_stride * tileY + thread;
		int tilepos = dep_stride * segLengthX + tileX - tt * isRegular;
	       	tilepos	+= thread/dep_stride * segLengthX + thread % dep_stride;
		intra_dep[pos] = tile[tilepos];
	}
}

//inter_stream_dep array structure: stream * tileT * dep_stride * (n1 + dep_stride)
__device__ void moveInterDepToTile(int* inter_stream_dep, int* tile, int tt, int tileX, int tileY, int dep_stride, int thread, int stream, int tileT, int n1, int segLengthX, int tileIdx, int len){
	int startAddress = (stream * tileT + tt) * dep_stride * (n1 + dep_stride);
	if (tileIdx > 0)       
		startAddress += ( (tileIdx-1) * tileX + tileX-tt );
	startAddress += ( tileIdx * tileX);
	//variable len specifies the eligible elements should be moved. This is caused by the irregular tile.
	if (thread < len + dep_stride){
		int pos = startAddress + thread;
		int tilepos = thread;
		for (int i=0; i<dep_stride; i++){
	 		tile[tilepos] = inter_stream_dep[pos];
			pos += (n1 + dep_stride);
			tilepos += segLengthX;
		}
	}	
}

__device__ void moveInterDepToTileEdge(volatile int* dev_arr, int* tile, int tileX, int tileY, int dep_stride, int thread, int n2, int segLengthX, int padd, int rowsize, int tileIdx, int tt, int len, int offset = 0){
	int glbpos = (padd - dep_stride) * rowsize + offset * (dep_stride + n2) * rowsize + padd - dep_stride + thread;
	if (tileIdx > 0)
		glbpos += ((tileIdx-1) * tileX + tileX-tt);
	if (thread < len + dep_stride){
		int tilepos = offset * (dep_stride + len) * segLengthX + thread;
		for (int i=0; i<dep_stride; i++){
			tile[tilepos] = dev_arr[glbpos];
			tilepos += segLengthX;
			glbpos += rowsize;
		}
	}
}

__device__ void moveTileToInterDep(int* inter_stream_dep, int* tile, int tt, int tileX, int tileY, int dep_stride, int thread, int nextSMStream, int tileT, int n1, int segLengthX, int tileIdx, int len, int isRegular){
	int startAddress = dep_stride + (nextSMStream * tileT + tt) * dep_stride * (n1 + dep_stride);
	//for the edge tiles, the size is irregular so that the start position of some tt timestamp are not times of tileX.
	if (tileIdx > 0)       
		startAddress += ( (tileIdx-1) * tileX + tileX-tt );
	//variable len specifies the eligible elements should be moved. This is caused by the irregular tile.
	if (thread < len){
		int pos = startAddress + thread;
		int tilepos = dep_stride + (tileY - (tt+1) * isRegular) * segLengthX + thread;
		for (int i=0; i<dep_stride; i++){
	 		inter_stream_dep[pos] = tile[tilepos];
			pos += (n1 + dep_stride);
			tilepos += segLengthX;
		}
	}	
}

__device__ void moveTileToInterDepEdge(volatile int* dev_arr, int* inter_stream_dep, int tt, int tileX, int tileY, int tileT, int nextSMStream, int dep_stride, int n1, int tileIdx, int rowsize, int curBatch, int padd, int thread){
	int startAddress = (nextSMStream * tileT + tt) * dep_stride * (n1 + dep_stride);
	int glbpos = padd * rowsize + curBatch * tileY * rowsize + (padd - dep_stride) + (tileY - dep_stride) * rowsize;
	if (thread < dep_stride){
		int interpos = startAddress + thread;
		int pos = glbpos + thread;
		for (int i=0; i<dep_stride; i++){
	 		inter_stream_dep[interpos] = dev_arr[pos];
			pos += rowsize;
			interpos += (n1 + dep_stride);
		}
	}	
}

__device__ void moveShareToGlobalEdge(int* tile, volatile int* dev_arr, int startPos, int ignLenX, int ignLenY, int tileX, int tileY, int dep_stride, int rowsize, int segLengthX, int thread){
	int xidx, yidx, glbPos, tilePos;
	for (int tid = thread; tid < tileX * tileY; tid += blockDim.x){
		xidx = tid % tileX;
		yidx = tid / tileX;
		if (xidx < tileX - ignLenX && yidx < tileY - ignLenY){
			glbPos = startPos + yidx * rowsize + xidx;
			tilePos = (dep_stride + yidx) * segLengthX + dep_stride + xidx;
			dev_arr[glbPos] = tile[tilePos];
		}
	}	
}	

__device__ void moveShareToGlobal(int* tile, volatile int* dev_arr, int startPos, int tileX, int tileY, int dep_stride, int rowsize, int segLengthX, int thread){
	int xidx, yidx, glbPos, tilePos;
	for (int tid = thread; tid < tileX * tileY; tid += blockDim.x){
		xidx = tid % tileX;
		yidx = tid / tileX;
		glbPos = startPos + yidx * rowsize + xidx;
		tilePos = (dep_stride + yidx) * segLengthX + dep_stride + xidx;
		dev_arr[glbPos] = tile[tilePos];
	}	
}	
	

/*
//need a global array which has size of the number of batches in each t. 
//Each stream check the corresponding element in this array to see if it is true; it is true only when the batch beneath it and in the 
//previous t is already completed.
//If it is true, change it to false and start the computation. At the end, change it back to true when computation is finished.
__device__ void read_batch_lock_for_time(int* dev_time_lock, int curBatch, int thread){
	if (thread == 0){
		while(dev_time_lock[curBatch] != 1){
		}
		dev_time_lock[curBatch] = 0;
	}
	__syncthreads();
}

__device__ void write_batch_lock_for_time(int* dev_time_lock, int curBatch, int thread){
	if (thread == 0){
		dev_time_lock[curBatch] = 1;
	}
	__synchthreads();
}
*/

//Similar to the lock array in nested loop study; create a 1-d array for the size of number of total rows. 
//A counter value is used for each row.
//Besides, we need to create such an array for each time stamp.
__device__ void read_tile_lock_for_batch(volatile int* dev_row_lock, int curBatch, int thread, int tileIdx, int YoverX, int xseg, int yseg, int timepiece){
	if (thread == 0){
		int limit = min(tileIdx + YoverX, xseg);
		while(dev_row_lock[timepiece * yseg + curBatch] < limit){
		}
		printf("curBatch: %d, tileIdx: %d, timepiece: %d, value: %d, limit: %d\n", curBatch, tileIdx, timepiece, dev_row_lock[timepiece*yseg+curBatch], limit);
	}
	__syncthreads();
}

__device__ void write_tile_lock_for_batch(volatile int* dev_row_lock, int curBatch, int thread, int yseg, int timepiece){
	if (thread == 0){
		dev_row_lock[timepiece * yseg + curBatch + 1] += 1;
//		printf("curBatch: %d, timepiece: %d, update to lock at: %d, value: %d\n", curBatch, timepiece, timepiece*yseg+curBatch+1, dev_row_lock[timepiece*yseg+curBatch+1]);
	}
	__syncthreads();
}

//__global__ void GPU_Tile(int stride, int tileX, int tileY, int curBatch, int batchStartAddress, int* dev_row_lock, int timepiece, int xseg, int yseg, int tileT){
__global__ void GPU_Tile(volatile int* dev_arr, int curBatch, int curStartAddress, int tileX, int tileY, int padd, int stride, int rowStartOffset, int rowsize, int colsize, int xseg, int yseg, int n1, int n2, int warpbatch, int curSMStream, int nextSMStream, int* inter_stream_dep, int inter_stream_dep_size, int tileT, int timepiece, int batchStartAddress, volatile int* dev_row_lock){ 
//We assume row size n1 is the multiple of 32 and can be completely divided by tileX.
//For each row, the first tile and the last tile are computed separately from the other tiles.
//size of the shared memory is determined by the GPU architecture.
//tileX is multiple times of 32 to maximize the cache read.		
#ifdef DEBUG
	if (threadIdx.x == 0){
		printf("This is curBatch: %d, batchStartAddress: %d\n", curBatch, batchStartAddress);
	}
	__syncthreads();
#endif
	//need two arrays: 1. tile raw data; 2. intra-stream dependence
	__shared__ int tile1[5120];
	__shared__ int tile2[5120];
	__shared__ int intra_dep[2047];

	int thread = threadIdx.x;
	int dep_stride = stride + 1;
	int segLengthX = tileX + dep_stride;
	int segLengthY = tileY + dep_stride;
	int tileIdx = 0;
	int xidx, yidx;
	int tilePos, newtilePos, glbPos;
	int tileAddress;
	int YoverX = tileY/tileX;	
//if this is the first batch of the current t tile, have to copy the related dependence data from global tile array into global inter-stream-dependence array.
//Challenges: when stream 0 is still working on one of the current t tiles but stream 2 already starts processing the first batch of the next t tiles. Copying the dependence data to arr[stream[0]] does not work.
//for the first and last batches, we need charactorized function to take care of the edge elements.

//***********************************************************************************************************************************
//	read_batch_lock_for_time(timepiece, curBatch);
//processing the first tile of each row, use the near-edge elements for the out-of-range dependence.
	//wait until it is safe to launch and execute the new batch.

	if (curBatch == 0){
	//for the first batch, use the near-edge elements for the out-of-range dependence.
		//when tile = 0, the calculated data which are outside the range are not copied to tile2, tile size is shrinking 
		//along T dimension. Out-of-range elements are used for dependent data.
		tileAddress = batchStartAddress + tileIdx * tileX;
		read_tile_lock_for_batch(dev_row_lock, curBatch, thread, tileIdx, YoverX, xseg, yseg, timepiece);
/*		moveMatrixToTile(dev_arr, &tile1[0], segLengthX, tileX, tileY, dep_stride, tileAddress, rowsize, warpbatch, thread);
		for (int tt=0; tt<tileT; tt++){
			moveIntraDepToTileEdge(dev_arr, &tile1[0], stride, rowsize, segLengthX, dep_stride, thread, tt, padd, n1, tileY, 0);
			moveInterDepToTileEdge(dev_arr, &tile1[0], tileX, tileY, dep_stride, thread, n2, segLengthX, padd, rowsize, tileIdx, tt, tileX);
			for (int tid = thread; tid < tileX * tileY; tid += blockDim.x){
				//out-of-range results should be ignored
				//because of the bias, xidx and yidx are the pos of new time elements.
				//thread % tileX and thread / tileX are pos of current cached elements.
				xidx = tid % tileX;
				yidx = tid / tileX;
			        //tilePos is the index of each element, to be calculated in the next timestamp. shifted left and up by 1.
				tilePos = (dep_stride-1) * segLengthX + (dep_stride - 1) + yidx * segLengthX + xidx;	
				//newtilePos is the index where the new calculated elements should be stored into the shared tile2 array.
				//NEED MODIFICATION BECAUSE newtilePos is not correct here because of the irregular tile size.
				newtilePos = tilePos;
				//when curBatch == 0, eligible tile size is reduced along the timestamp because of the shifting.
				//Because the edge elements use only the out-of-range elements as dependent data, we need specific manipulation.
				if (xidx > 0 && xidx < tileX-tt && yidx > 0 && yidx < tileY-tt)
					tile2[newtilePos] = (tile1[tilePos+stride] + tile1[tilePos+segLengthX] + tile1[tilePos] + tile1[tilePos-stride] + tile1[tilePos-segLengthX]) / 5;
			}	
			__syncthreads();
			
			//Since the tile size is reduced along the calculation, the intraDep elements (in last two column of the valid tile) is also shifted to left.
			//Set variable isRegular == 1, when there is a size reduction. 
			moveTileToIntraDep(&intra_dep[0], &tile1[0], tt, tileX, tileY, segLengthX, dep_stride, thread, 1, tileY);
			//first tile has to copy the out-of-range elements, which are on the left-hand side, to next stream's inter_stream_dep array
			moveTileToInterDepEdge(dev_arr, inter_stream_dep, tt, tileX, tileY, tileT, nextSMStream, dep_stride, n1, tileIdx, rowsize, curBatch, padd, thread);
			//variable isRegular == 1, because one row is shifted out-side-of the upper boundary
			//variable len == tileX-tt because this tile is not in a regular size.
			moveTileToInterDep(&inter_stream_dep[0], &tile1[0], tt, tileX, tileY, dep_stride, thread, nextSMStream, tileT, n1, segLengthX, tileIdx, tileX-tt, 1);
			//swap tile2 with tile1;
			for (int tid = thread; tid < 5120; tid+=blockDim.x){
				tile1[tid] = tile2[tid];
				tile2[tid] = 0;
			}
			__syncthreads();
		}
		//glbPos is the index where the calculated elements should be stored at in the global matrix array.
		//when curBatch == 0 && tileIdx == 0, glbPos always start from the first eligible element of the tile, which is tileAddress
		//and then ignore the out-of-range elements by using ignLenX and ignLenY variables.
		//when curBatch == 0 or tile idx == 0, the out of range elements should be ignored, ignLenX and ignLenY are set accordingly.
		//curBatch > 0 && tileIdx == 0, glbPos is shifted up by tileT unit from tileAddress.
		//curBatch == 0 && tileIdx > 0, glbPos is shifted left by tileT unit from tileAddress.
		//when curBatch > 0 && tileIdx > 0, glbPos is shifted up and left by tileT unit from tileAddress, complete tile is moved,
		//ignLenX == tileT because tileX-tileT elements are copied at each row, ignLenY == tileT because tileY-tileT elements are copied at each column.
		glbPos = tileAddress;	
		moveShareToGlobalEdge(&tile1[0], dev_arr, glbPos, tileT, tileT, tileX, tileY, dep_stride, rowsize, segLengthX, thread);	
		__syncthreads();
*/		write_tile_lock_for_batch(dev_row_lock, curBatch, thread, yseg, timepiece);

		//tile = 1 to xseg-1; regular size tiles, with index shifting.
		for (tileIdx = 1; tileIdx < xseg-1; tileIdx++){
			tileAddress = batchStartAddress + tileIdx * tileX;
			read_tile_lock_for_batch(dev_row_lock, curBatch, thread, tileIdx, YoverX, xseg, yseg, timepiece);
/*			//copy the base spatial data to shared memory for t=0.
			moveMatrixToTile(dev_arr, &tile1[0], segLengthX, tileX, tileY, dep_stride, tileAddress, rowsize, warpbatch, thread);
			for (int tt=0; tt<tileT; tt++){
				moveIntraDepToTile(&intra_dep[0], &tile1[0], tt, tileY, segLengthX, dep_stride, thread, tileY);
				moveInterDepToTileEdge(dev_arr, &tile1[0], tileX, tileY, dep_stride, thread, n2, segLengthX, padd, rowsize, tileIdx, tt, tileX);
				for (int tid = thread; tid < tileX * tileY; tid += blockDim.x){
					//out-of-range results should be ignored
					//because of the bias, xidx and yidx are the pos of new time elements.
					//thread % tileX and thread / tileX are pos of current cached elements.
					xidx = tid % tileX;
					yidx = tid / tileX;
				        //tilePos is the index of each element, to be calculated in the next timestamp. shifted left and up by 1
					tilePos = (dep_stride-1) * segLengthX + (dep_stride - 1) + yidx * segLengthX + xidx;	
					//newtilePos is the index where the new calculated elements should be stored into the shared tile2 array
					//newtilePos = dep_stride * segLengthX + dep_stride + yidx * segLengthX + xidx;
					newtilePos = tilePos;
					//when curBatch == 0, eligible tile size is reduced along the timestamp because of the shifting.
					//Because, the edge elements use only the out-of-range elements as dependent data, we need specific manipulation.
					if (yidx > 0 && yidx < tileX-tt)
						tile2[newtilePos] = (tile1[tilePos+stride] + tile1[tilePos+segLengthX] + tile1[tilePos] + tile1[tilePos-stride] + tile1[tilePos-segLengthX]) / 5;
				}	
				__syncthreads();
				//Set variable isRegular == 0 to disable the tile size reduction, when tile size are constant during the calculation. 
				moveTileToIntraDep(&intra_dep[0], &tile1[0], tt, tileX, tileY, segLengthX, dep_stride, thread, 0, tileY);
				//variable isRegular == 1 because one row is shifted out-side-of the upper boundary.
				//variable len == tileX-tt because row is shifted out-side-of the upper boundary
				moveTileToInterDep(&inter_stream_dep[0], &tile1[0], tt, tileX, tileY, dep_stride, thread, nextSMStream, tileT, n1, segLengthX, tileIdx, tileX-tt, 1);
				//swap tile2 with tile1;
				for (int tid = thread; tid < 5120; tid+=blockDim.x){
					tile1[tid] = tile2[tid];
					tile2[tid] = 0;
				}
				__syncthreads();
			}						 
			//glbPos is the index where the calculated elements should be stored at in the global matrix array.
			//when curBatch == 0 && tileIdx == 0, glbPos always start from the first eligible element of the tile, which is tileAddress
			//and then ignore the out-of-range elements by using ignLenX and ignLenY variables.
			//when curBatch == 0 or tile idx == 0, the out of range elements should be ignored, ignLenX and ignLenY are set accordingly.
			//curBatch > 0 && tileIdx == 0, glbPos is shifted up by tileT unit from tileAddress.
			//curBatch == 0 && tileIdx > 0, glbPos is shifted left by tileT unit from tileAddress.
			//when curBatch > 0 && tileIdx > 0, glbPos is shifted up and left by tileT unit from tileAddress, complete tile is moved,
			//ignLenX == 0 because all elements are copied at each row, ignLenY == tileT because tileY-tileT elements are copied at each column.
			glbPos = tileAddress;	
			moveShareToGlobalEdge(&tile1[0], dev_arr, glbPos, 0, tileT, tileX, tileY, dep_stride, rowsize, segLengthX, thread);	
			__syncthreads();
*/	
			write_tile_lock_for_batch(dev_row_lock, curBatch, thread, yseg, timepiece);
		}

		//when tile = xseg-1, if matrix is completely divided by the tile, no t0 elements copy to shared memory; 
		//use dependent data and out-of-range data to calculate.
		tileIdx = xseg-1;
		//unlike the other two cases that tileAddress points to the source pos of t0, here tileAddress is the destination pos of t(tileT-1).
		tileAddress = batchStartAddress + tileIdx * tileX - tileT;
		read_tile_lock_for_batch(dev_row_lock, curBatch, thread, tileIdx, YoverX, xseg, yseg, timepiece);
/*		for (int tt=0; tt<tileT; tt++){
			moveIntraDepToTile(&intra_dep[0], &tile1[0], tt, tileY, segLengthX, dep_stride, thread, tileY);
			//set variable offset == 1 if it is the last tile of each batch to copy right-side out-of-range elements to 
			moveIntraDepToTileEdge(dev_arr, &tile1[0], stride, rowsize, segLengthX, dep_stride, thread, tt, padd, n1, tileY, 1);
			moveInterDepToTileEdge(dev_arr, &tile1[0], tileX, tileY, dep_stride, thread, n2, segLengthX, padd, rowsize, tileIdx, tt, tt + dep_stride);
			//tileX of the last tile is changed throughout the simulation from 0 to tileT;
			for (int tid = thread; tid < (tt+1) * tileY; tid += blockDim.x){
				//out-of-range results should be ignored
				//because of the bias, xidx and yidx are the pos of new time elements.
				//thread % tileX and thread / tileX are pos of current cached elements.
				xidx = tid % tileX;
				yidx = tid / tileX;
			        //tilePos is the index of each element, to be calculated in the next timestamp. shifted left and up by 1.
				tilePos = (dep_stride-1) * segLengthX + (dep_stride - 1) + yidx * segLengthX + xidx;	
				//newtilePos starts one row above the tile matrix because the next tile is shifted out-side-of the up boundary
				newtilePos = (dep_stride-1) * segLengthX + dep_stride + yidx * segLengthX + xidx;
				//when curBatch == 0, eligible tile size is reduced along the timestamp because of the shifting.
				//Because, the edge elements use only the out-of-range elements as dependent data, we need specific manipulation
				if (xidx <= tt && yidx > 0 && yidx < tileY-tt)
					tile2[newtilePos] = (tile1[tilePos+stride] + tile1[tilePos+segLengthX] + tile1[tilePos] + tile1[tilePos-stride] + tile1[tilePos-segLengthX]) / 5;
			}	
			__syncthreads();
			
			//variable isRegular == 1 because one row is shifted out-side-of the upper boundary.
			//len = tileX-1-tt, variable len specifies the lenth of eligible elements should be moved to inter_stream_dep[].
			moveTileToInterDep(&inter_stream_dep[0], &tile1[0], tt, tileX, tileY, dep_stride, thread, nextSMStream, tileT, n1, segLengthX, tileIdx, tileX-tt-1, 1);
			//swap tile2 with tile1;
			for (int tid = thread; tid < 5120; tid+=blockDim.x){
				tile1[tid] = tile2[tid];
				tile2[tid] = 0;
			}
			__syncthreads();
		}
		//glbPos is the index where the calculated elements should be stored at in the global matrix array.
		//when curBatch == 0 && tileIdx == 0, glbPos always start from the first eligible element of the tile, which is tileAddress
		//and then ignore the out-of-range elements by using ignLenX and ignLenY variables.
		//when curBatch == 0 or tile idx == 0, the out of range elements should be ignored, ignLenX and ignLenY are set accordingly.
		//curBatch > 0 && tileIdx == 0, glbPos is shifted up by tileT unit from tileAddress.
		//curBatch == 0 && tileIdx > 0, glbPos is shifted left by tileT unit from tileAddress.
		//when curBatch > 0 && tileIdx > 0, glbPos is shifted up and left by tileT unit from tileAddress, complete tile is moved,
		//ignLenX == tileX-tileT because tileT elements are copied at each row, ignLenY == tileT because tileY-tileT elements are copied at each column.
		glbPos = tileAddress;	
		moveShareToGlobalEdge(&tile1[0], dev_arr, glbPos, tileX-tileT, tileT, tileX, tileY, dep_stride, rowsize, segLengthX, thread);	
		__syncthreads();
*/
		write_tile_lock_for_batch(dev_row_lock, curBatch, thread, yseg, timepiece);

	}
	else if(curBatch == yseg - 1){
	//for the last batch, all the tiles are irregular
		//when tile = 0, the calculated data which are outside the range are not copied to tile2, tile size is shrinking 
		//along T dimension. Out-of-range elements are used for dependent data.
		read_tile_lock_for_batch(dev_row_lock, curBatch, thread, tileIdx, YoverX, xseg, yseg, timepiece);
/*		for (int tt=0; tt<tileT; tt++){
			moveIntraDepToTileEdge(dev_arr, &tile1[0], stride, rowsize, segLengthX, dep_stride, thread, tt, padd, n1, tt, 1);
			//the first tile is not in regular size, so variable len = tileX-tt
			moveInterDepToTile(inter_stream_dep, &tile1[0], tt, tileX, tileY, dep_stride, thread, curSMStream, tileT, n1, segLengthX, tileIdx, tileX-tt);
			//move out-of-range elements which are beanth the bottom boundary to the tile
			//variable offset == 1, used to locate the bottom out-of-boundary elements.
			moveInterDepToTileEdge(dev_arr, &tile1[0], tileX, tileY, dep_stride, thread, n2, segLengthX, padd, rowsize, tileIdx, tt, tileX, 1);
			
			for (int tid = thread; tid < tileX * tileY; tid += blockDim.x){
				//out-of-range results should be ignored
				//because of the bias, xidx and yidx are the pos of new time elements.
				//thread % tileX and thread / tileX are pos of current cached elements.
				xidx = tid % tileX;
				yidx = tid / tileX;
			        //tilePos is the index of each element, to be calculated in the next timestamp. shifted left and up by 1.
				tilePos = (dep_stride-1) * segLengthX + (dep_stride - 1) + yidx * segLengthX + xidx;	
				//newtilePos is the index where the new calculated elements should be stored into the shared tile2 array.
				//left column shift out-side-of the boundary, so retain all rows but discard the left-most column.
				newtilePos = dep_stride * segLengthX + (dep_stride - 1) + yidx * segLengthX + xidx;
				//when curBatch == 0, eligible tile size is reduced along the timestamp because of the shifting.
				//Because the edge elements use only the out-of-range elements as dependent data, we need specific manipulation.
				if (xidx>0 && xidx < tileX-tt && yidx <= tt)
					tile2[newtilePos] = (tile1[tilePos+stride] + tile1[tilePos+segLengthX] + tile1[tilePos] + tile1[tilePos-stride] + tile1[tilePos-segLengthX]) / 5;
			}	
			__syncthreads();
			
			//Since the tile size is reduced along the calculation, the intraDep elements (in last two column of the valid tile) is also shifted to left.
			//Set variable isRegular == 1, when there is a size reduction. 
			moveTileToIntraDep(&intra_dep[0], &tile1[0], tt, tileX, tileY, segLengthX, dep_stride, thread, 1, tt);
			//swap tile2 with tile1;
			for (int tid = thread; tid < 5120; tid+=blockDim.x){
				tile1[tid] = tile2[tid];
				tile2[tid] = 0;
			}
			__syncthreads();
		}
		//glbPos is the index where the calculated elements should be stored at in the global matrix array.
		//when curBatch == 0 && tileIdx == 0, glbPos always start from the first eligible element of the tile, which is tileAddress
		//and then ignore the out-of-range elements by using ignLenX and ignLenY variables.
		//when curBatch == 0 or tile idx == 0, the out of range elements should be ignored, ignLenX and ignLenY are set accordingly.
		//curBatch > 0 && tileIdx == 0, glbPos is shifted up by tileT unit from tileAddress.
		//curBatch == 0 && tileIdx > 0, glbPos is shifted left by tileT unit from tileAddress.
		//when curBatch > 0 && tileIdx > 0, glbPos is shifted up and left by tileT unit from tileAddress, complete tile is moved,
		glbPos = batchStartAddress + tileIdx * tileX;
		//ignLenX == tileT because tileX-tileT elements are copied at each row, ignLenY == tileY-tileT because tileT elements are copied at each column.
		moveShareToGlobalEdge(&tile1[0], dev_arr, glbPos, tileT, tileY-tileT, tileX, tileY, dep_stride, rowsize, segLengthX, thread);	
		__syncthreads();
*/
//		write_tile_lock_for_batch(dev_row_lock, curBatch, thread, yseg, timepiece);

		//tile = 1 to xseg-1; regular size tiles, with index shifting.
		for (tileIdx = 1; tileIdx < xseg-1; tileIdx++){
			read_tile_lock_for_batch(dev_row_lock, curBatch, thread, tileIdx, YoverX, xseg, yseg, timepiece);
/*			for (int tt=0; tt<tileT; tt++){
				//isRegular == 0 because this is a regular tile.
//				moveIntraDepToTile(&intra_dep[0], &tile1[0], tt, tileY, segLengthX, dep_stride, thread, 0, tt);
				moveIntraDepToTile(&intra_dep[0], &tile1[0], tt, tileY, segLengthX, dep_stride, thread, tt);
				moveInterDepToTile(inter_stream_dep, &tile1[0], tt, tileX, tileY, dep_stride, thread, curSMStream, tileT, n1, segLengthX, tileIdx, tileX);
				//move out-of-range elements which are beanth the bottom boundary to the tile
				//variable offset == 1, used to locate the bottom out-of-boundary elements.
				moveInterDepToTileEdge(dev_arr, &tile1[0], tileX, tileY, dep_stride, thread, n2, segLengthX, padd, rowsize, tileIdx, tt, tileX, 1);
				for (int tid = thread; tid < tileX * tileY; tid += blockDim.x){
					//out-of-range results should be ignored
					//because of the bias, xidx and yidx are the pos of new time elements.
					//thread % tileX and thread / tileX are pos of current cached elements.
					xidx = tid % tileX;
					yidx = tid / tileX;
				        //tilePos is the index of each element, to be calculated in the next timestamp. shifted left and up by 1
					tilePos = (dep_stride-1) * segLengthX + (dep_stride - 1) + yidx * segLengthX + xidx;	
					//newtilePos is the index where the new calculated elements should be stored into the shared tile2 array
					newtilePos = dep_stride * segLengthX + dep_stride + yidx * segLengthX + xidx;
					//when curBatch == 0, eligible tile size is reduced along the timestamp because of the shifting.
					//Because, the edge elements use only the out-of-range elements as dependent data, we need specific manipulation.
					if (yidx <= tt)
						tile2[newtilePos] = (tile1[tilePos+stride] + tile1[tilePos+segLengthX] + tile1[tilePos] + tile1[tilePos-stride] + tile1[tilePos-segLengthX]) / 5;
				}	
				__syncthreads();
				//isRegular == 0 to disable the tile size reduction, when tile size are constant during the calculation. 
				moveTileToIntraDep(&intra_dep[0], &tile1[0], tt, tileX, tileY, segLengthX, dep_stride, thread, 0, tt);
				//variable len == tileX because the tile size is constant.
				moveTileToInterDep(&inter_stream_dep[0], &tile1[0], tt, tileX, tileY, dep_stride, thread, nextSMStream, tileT, n1, segLengthX, tileIdx, tileX, 0);
				//swap tile2 with tile1;
				for (int tid = thread; tid < 5120; tid+=blockDim.x){
					tile1[tid] = tile2[tid];
					tile2[tid] = 0;
				}
				__syncthreads();
			}						 
			//glbPos is the index where the calculated elements should be stored at in the global matrix array.
			//when curBatch == 0 && tileIdx == 0, glbPos always start from the first eligible element of the tile, which is tileAddress
			//and then ignore the out-of-range elements by using ignLenX and ignLenY variables.
			//when curBatch == 0 or tile idx == 0, the out of range elements should be ignored, ignLenX and ignLenY are set accordingly.
			//curBatch > 0 && tileIdx == 0, glbPos is shifted up by tileT unit from tileAddress.
			//curBatch == 0 && tileIdx > 0, glbPos is shifted left by tileT unit from tileAddress.
			//when curBatch > 0 && tileIdx > 0, glbPos is shifted up and left by tileT unit from tileAddress, complete tile is moved,
			glbPos = batchStartAddress + tileIdx * tileX;
			//ignLenX == 0 because all elements are copied at each row, ignLenY == tileY-tileT because tileT elements are copied at each column.
			moveShareToGlobalEdge(&tile1[0], dev_arr, glbPos, 0, tileY-tileT, tileX, tileY, dep_stride, rowsize, segLengthX, thread);	
			__syncthreads();
			
*/
//			write_tile_lock_for_batch(dev_row_lock, curBatch, thread, yseg, timepiece);
		}

		//when tile = xseg-1, if matrix is completely divided by the tile, no t0 elements copy to shared memory; 
		//use dependent data and out-of-range data to calculate.
		tileIdx = xseg-1;
		read_tile_lock_for_batch(dev_row_lock, curBatch, thread, tileIdx, YoverX, xseg, yseg, timepiece);
/*		for (int tt=0; tt<tileT; tt++){
			moveIntraDepToTile(&intra_dep[0], &tile1[0], tt, tileY, segLengthX, dep_stride, thread, tt);
			//set variable offset == 1 if it is the last tile of each batch to copy right-side out-of-range elements to 
			moveIntraDepToTileEdge(dev_arr, &tile1[0], stride, rowsize, segLengthX, dep_stride, thread, tt, padd, n1, tt, 1);
			
			//1. inter_stream_dep elements from previous tile (on top of intra_dep elements); total size == len + dev_stride, where len == tt, which is 0 at t0
			//2. out-of-range elements
			//copy edge elements first to cover the out-of-range elements, then copy the inter_stream_dep of previous stream and cover a part of the out-of-range elements.
			//variable len == tt + dev_stride, which covers the size of the elements, calculated in previous stream, and the out-of-range elements.
			moveInterDepToTileEdge(dev_arr, &tile1[0], tileX, tileY, dep_stride, thread, n2, segLengthX, padd, rowsize, tileIdx, tt, tt + dep_stride);
			moveInterDepToTile(inter_stream_dep, &tile1[0], tt, tileX, tileY, dep_stride, thread, curSMStream, tileT, n1, segLengthX, tileIdx, tt);
			//move out-of-range elements which are beanth the bottom boundary to the tile
			//variable offset == 1, used to locate the bottom out-of-boundary elements.
			moveInterDepToTileEdge(dev_arr, &tile1[0], tileX, tileY, dep_stride, thread, n2, segLengthX, padd, rowsize, tileIdx, tt, tt + dep_stride, 1);
			//tileX of the last tile is changed throughout the simulation from 0 to tileT;
			for (int tid = thread; tid < (tt+1) * tileY; tid += blockDim.x){
				//out-of-range results should be ignored
				//because of the bias, xidx and yidx are the pos of new time elements.
				//thread % tileX and thread / tileX are pos of current cached elements.
				xidx = tid % tileX;
				yidx = tid / tileX;
			        //tilePos is the index of each element, to be calculated in the next timestamp. shifted left and up by 1.
				tilePos = (dep_stride-1) * segLengthX + (dep_stride - 1) + yidx * segLengthX + xidx;	
				//newtilePos is the index where the new calculated elements should be stored into the shared tile2 array.
				newtilePos = dep_stride * segLengthX + dep_stride + yidx * segLengthX + xidx;
				//when curBatch == 0, eligible tile size is reduced along the timestamp because of the shifting.
				//Because, the edge elements use only the out-of-range elements as dependent data, we need specific manipulation
				if (xidx <= tt && yidx <= tt)
					tile2[newtilePos] = (tile1[tilePos+stride] + tile1[tilePos+segLengthX] + tile1[tilePos] + tile1[tilePos-stride] + tile1[tilePos-segLengthX]) / 5;
			}	
			__syncthreads();
			
			//swap tile2 with tile1;
			for (int tid = thread; tid < 5120; tid+=blockDim.x){
				tile1[tid] = tile2[tid];
				tile2[tid] = 0;
			}
			__syncthreads();
		}
		//glbPos is the index where the calculated elements should be stored at in the global matrix array.
		//when curBatch == 0 && tileIdx == 0, glbPos always start from the first eligible element of the tile, which is tileAddress
		//and then ignore the out-of-range elements by using ignLenX and ignLenY variables.
		//when curBatch == 0 or tile idx == 0, the out of range elements should be ignored, ignLenX and ignLenY are set accordingly.
		//curBatch > 0 && tileIdx == 0, glbPos is shifted up by tileT unit from tileAddress.
		//curBatch == 0 && tileIdx > 0, glbPos is shifted left by tileT unit from tileAddress.
		//when curBatch > 0 && tileIdx > 0, glbPos is shifted up and left by tileT unit from tileAddress, complete tile is moved,
		//unlike the other two cases that glbPos points to the source pos of t0, here tileAddress is the destination pos of t(tileT-1).
		glbPos = batchStartAddress + tileIdx * tileX - tileT;	
		//ignLenX == tileX-tileT because tileT elements are copied at each row, ignLenY == tileY-tileT because tileT elements are copied at each column.
		moveShareToGlobalEdge(&tile1[0], dev_arr, glbPos, tileX-tileT, tileY-tileT, tileX, tileY, dep_stride, rowsize, segLengthX, thread);	
		__syncthreads();
*/	
//		write_tile_lock_for_batch(dev_row_lock, curBatch, thread, yseg, timepiece);
	
	}
	else{
	//for the regular batch, use the near-edge elements for the out-of-range dependence of first and last tile only.
		//when tile = 0, the calculated data which are outside the range are not copied to tile2, tile size is shrinking 
		//along T dimension. Out-of-range elements are used for dependent data.
		tileAddress = batchStartAddress + tileIdx * tileX;
		read_tile_lock_for_batch(dev_row_lock, curBatch, thread, tileIdx, YoverX, xseg, yseg, timepiece);
/*		moveMatrixToTile(dev_arr, &tile1[0], segLengthX, tileX, tileY, dep_stride, tileAddress, rowsize, warpbatch, thread);
		for (int tt=0; tt<tileT; tt++){
			moveIntraDepToTileEdge(dev_arr, &tile1[0], stride, rowsize, segLengthX, dep_stride, thread, tt, padd, n1, tileY, 0);
			//the first tile is not in regular size, so variable len = tileX-tt
			moveInterDepToTile(inter_stream_dep, &tile1[0], tt, tileX, tileY, dep_stride, thread, curSMStream, tileT, n1, segLengthX, tileIdx, tileX-tt);
			for (int tid = thread; tid < tileX * tileY; tid += blockDim.x){
				//out-of-range results should be ignored
				//because of the bias, xidx and yidx are the pos of new time elements.
				//thread % tileX and thread / tileX are pos of current cached elements.
				xidx = tid % tileX;
				yidx = tid / tileX;
			        //tilePos is the index of each element, to be calculated in the next timestamp. shifted left and up by 1.
				tilePos = (dep_stride-1) * segLengthX + (dep_stride - 1) + yidx * segLengthX + xidx;	
				//newtilePos is the index where the new calculated elements should be stored into the shared tile2 array.
				//left column shift out-side-of the boundary, so retain all rows but discard the left-most column.
				newtilePos = dep_stride * segLengthX + (dep_stride - 1) + yidx * segLengthX + xidx;
				//when curBatch == 0, eligible tile size is reduced along the timestamp because of the shifting.
				//Because the edge elements use only the out-of-range elements as dependent data, we need specific manipulation.
				if (xidx > 0 && xidx < tileX-tt)
					tile2[newtilePos] = (tile1[tilePos+stride] + tile1[tilePos+segLengthX] + tile1[tilePos] + tile1[tilePos-stride] + tile1[tilePos-segLengthX]) / 5;
			}	
			__syncthreads();
			
			//Since the tile size is reduced along the calculation, the intraDep elements (in last two column of the valid tile) is also shifted to left.
			//Set variable isRegular == 1, when there is a size reduction. 
			moveTileToIntraDep(&intra_dep[0], &tile1[0], tt, tileX, tileY, segLengthX, dep_stride, thread, 1, tileY);
			//first tile has to copy the out-of-range elements, which are on the left-hand side, to next stream's inter_stream_dep array
			moveTileToInterDepEdge(dev_arr, inter_stream_dep, tt, tileX, tileY, tileT, nextSMStream, dep_stride, n1, tileIdx, rowsize, curBatch, padd, thread);
			//variable len == tileX-tt because the tile size is reduced during calculation.
			//isRegular == 0 because there is no row move out-side-of upper boundary
			moveTileToInterDep(&inter_stream_dep[0], &tile1[0], tt, tileX, tileY, dep_stride, thread, nextSMStream, tileT, n1, segLengthX, tileIdx, tileX-tt, 0);
			//swap tile2 with tile1;
			for (int tid = thread; tid < 5120; tid+=blockDim.x){
				tile1[tid] = tile2[tid];
				tile2[tid] = 0;
			}
			__syncthreads();
		}
		//glbPos is the index where the calculated elements should be stored at in the global matrix array.
		//when curBatch == 0 && tileIdx == 0, glbPos always start from the first eligible element of the tile, which is tileAddress
		//and then ignore the out-of-range elements by using ignLenX and ignLenY variables.
		//when curBatch == 0 or tile idx == 0, the out of range elements should be ignored, ignLenX and ignLenY are set accordingly.
		//curBatch > 0 && tileIdx == 0, glbPos is shifted up by tileT unit from tileAddress.
		//curBatch == 0 && tileIdx > 0, glbPos is shifted left by tileT unit from tileAddress.
		//when curBatch > 0 && tileIdx > 0, glbPos is shifted up and left by tileT unit from tileAddress, complete tile is moved,
		//ignLenX == tileT because tileX-tileT elements are copied at each row, ignLenY == 0 because no size reduction along Y dim.
		glbPos = tileAddress;	
		moveShareToGlobalEdge(&tile1[0], dev_arr, glbPos, tileT, 0, tileX, tileY, dep_stride, rowsize, segLengthX, thread);	
		__syncthreads();
*/		write_tile_lock_for_batch(dev_row_lock, curBatch, thread, yseg, timepiece);

		//tile = 1 to xseg-1; regular size tiles, with index shifting.
		for (tileIdx = 1; tileIdx < xseg-1; tileIdx++){
			tileAddress = batchStartAddress + tileIdx * tileX;
			read_tile_lock_for_batch(dev_row_lock, curBatch, thread, tileIdx, YoverX, xseg, yseg, timepiece);
/*			//copy the base spatial data to shared memory for t=0.
			moveMatrixToTile(dev_arr, &tile1[0], segLengthX, tileX, tileY, dep_stride, tileAddress, rowsize, warpbatch, thread);
			for (int tt=0; tt<tileT; tt++){
				//isRegular == 0 because this is a regular tile.
//				moveIntraDepToTile(&intra_dep[0], &tile1[0], tt, tileY, segLengthX, dep_stride, thread, 0, tileY);
				moveIntraDepToTile(&intra_dep[0], &tile1[0], tt, tileY, segLengthX, dep_stride, thread, tileY);
				moveInterDepToTile(inter_stream_dep, &tile1[0], tt, tileX, tileY, dep_stride, thread, curSMStream, tileT, n1, segLengthX, tileIdx, tileX);
				for (int tid = thread; tid < tileX * tileY; tid += blockDim.x){
					//out-of-range results should be ignored
					//because of the bias, xidx and yidx are the pos of new time elements.
					//thread % tileX and thread / tileX are pos of current cached elements.
					xidx = tid % tileX;
					yidx = tid / tileX;
				        //tilePos is the index of each element, to be calculated in the next timestamp. shifted left and up by 1
					tilePos = (dep_stride-1) * segLengthX + (dep_stride - 1) + yidx * segLengthX + xidx;	
					//newtilePos is the index where the new calculated elements should be stored into the shared tile2 array
					newtilePos = dep_stride * segLengthX + dep_stride + yidx * segLengthX + xidx;
					tile2[newtilePos] = (tile1[tilePos+stride] + tile1[tilePos+segLengthX] + tile1[tilePos] + tile1[tilePos-stride] + tile1[tilePos-segLengthX]) / 5;
				}	
				__syncthreads();
				//isRegular == 0 to disable the tile size reduction, when tile size are constant during the calculation. 
				moveTileToIntraDep(&intra_dep[0], &tile1[0], tt, tileX, tileY, segLengthX, dep_stride, thread, 0, tileY);
				//variable len == tileX because the tile size is constant.
				moveTileToInterDep(&inter_stream_dep[0], &tile1[0], tt, tileX, tileY, dep_stride, thread, nextSMStream, tileT, n1, segLengthX, tileIdx, tileX, 0);
				//swap tile2 with tile1;
				for (int tid = thread; tid < 5120; tid+=blockDim.x){
					tile1[tid] = tile2[tid];
					tile2[tid] = 0;
				}
				__syncthreads();
			}						 
			//glbPos is the index where the calculated elements should be stored at in the global matrix array.
			//when curBatch == 0 && tileIdx == 0, glbPos always start from the first eligible element of the tile, which is tileAddress
			//and then ignore the out-of-range elements by using ignLenX and ignLenY variables.
			//when curBatch == 0 or tile idx == 0, the out of range elements should be ignored, ignLenX and ignLenY are set accordingly.
			//curBatch > 0 && tileIdx == 0, glbPos is shifted up by tileT unit from tileAddress.
			//curBatch == 0 && tileIdx > 0, glbPos is shifted left by tileT unit from tileAddress.
			//when curBatch > 0 && tileIdx > 0, glbPos is shifted up and left by tileT unit from tileAddress, complete tile is moved,
			glbPos = tileAddress;	
			moveShareToGlobal(&tile1[0], dev_arr, glbPos, tileX, tileY, dep_stride, rowsize, segLengthX, thread);	
			__syncthreads();
*/			
			write_tile_lock_for_batch(dev_row_lock, curBatch, thread, yseg, timepiece);
		}

		//when tile = xseg-1, if matrix is completely divided by the tile, no t0 elements copy to shared memory; 
		//use dependent data and out-of-range data to calculate.
		tileIdx = xseg-1;
		//unlike the other two cases that tileAddress points to the source pos of t0, here tileAddress is the destination pos of t(tileT-1).
		tileAddress = batchStartAddress + tileIdx * tileX - tileT;
		read_tile_lock_for_batch(dev_row_lock, curBatch, thread, tileIdx, YoverX, xseg, yseg, timepiece);
/*		for (int tt=0; tt<tileT; tt++){
			moveIntraDepToTile(&intra_dep[0], &tile1[0], tt, tileY, segLengthX, dep_stride, thread, tileY);
			//set variable offset == 1 if it is the last tile of each batch to copy right-side out-of-range elements to 
			moveIntraDepToTileEdge(dev_arr, &tile1[0], stride, rowsize, segLengthX, dep_stride, thread, tt, padd, n1, tileY, 1);
			
			//1. inter_stream_dep elements from previous tile (on top of intra_dep elements); total size == len + dev_stride, where len == tt, which is 0 at t0
			//2. out-of-range elements
			//copy edge elements first to cover the out-of-range elements, then copy the inter_stream_dep of previous stream and cover a part of the out-of-range elements.
			//variable len == tt + dev_stride, which covers the size of the elements, calculated in previous stream, and the out-of-range elements.
			moveInterDepToTileEdge(dev_arr, &tile1[0], tileX, tileY, dep_stride, thread, n2, segLengthX, padd, rowsize, tileIdx, tt, tt + dep_stride);
			moveInterDepToTile(inter_stream_dep, &tile1[0], tt, tileX, tileY, dep_stride, thread, curSMStream, tileT, n1, segLengthX, tileIdx, tt);
			//tileX of the last tile is changed throughout the simulation from 0 to tileT;
			for (int tid = thread; tid < (tt+1) * tileY; tid += blockDim.x){
				//out-of-range results should be ignored
				//because of the bias, xidx and yidx are the pos of new time elements.
				//thread % tileX and thread / tileX are pos of current cached elements.
				xidx = tid % tileX;
				yidx = tid / tileX;
			        //tilePos is the index of each element, to be calculated in the next timestamp. shifted left and up by 1.
				tilePos = (dep_stride-1) * segLengthX + (dep_stride - 1) + yidx * segLengthX + xidx;	
				//newtilePos is the index where the new calculated elements should be stored into the shared tile2 array.
				newtilePos = dep_stride * segLengthX + dep_stride + yidx * segLengthX + xidx;
				//when curBatch == 0, eligible tile size is reduced along the timestamp because of the shifting.
				//Because, the edge elements use only the out-of-range elements as dependent data, we need specific manipulation
				if (xidx <= tt)
					tile2[newtilePos] = (tile1[tilePos+stride] + tile1[tilePos+segLengthX] + tile1[tilePos] + tile1[tilePos-stride] + tile1[tilePos-segLengthX]) / 5;
			}	
			__syncthreads();
			
			//variable isRegular == 0 because one row is shifted out-side-of the upper boundary.
			//len = tileX-1-tt, variable len specifies the lenth of eligible elements should be moved to inter_stream_dep[].
			moveTileToInterDep(&inter_stream_dep[0], &tile1[0], tt, tileX, tileY, dep_stride, thread, nextSMStream, tileT, n1, segLengthX, tileIdx, tileX-tt-1, 0);
			//swap tile2 with tile1;
			}
			__syncthreads();
		}
		//glbPos is the index where the calculated elements should be stored at in the global matrix array.
		//when curBatch == 0 && tileIdx == 0, glbPos always start from the first eligible element of the tile, which is tileAddress
		//and then ignore the out-of-range elements by using ignLenX and ignLenY variables.
		//when curBatch == 0 or tile idx == 0, the out of range elements should be ignored, ignLenX and ignLenY are set accordingly.
		//curBatch > 0 && tileIdx == 0, glbPos is shifted up by tileT unit from tileAddress.
		//curBatch == 0 && tileIdx > 0, glbPos is shifted left by tileT unit from tileAddress.
		//when curBatch > 0 && tileIdx > 0, glbPos is shifted up and left by tileT unit from tileAddress, complete tile is moved,
		//ignLenX == tileX-tileT because only tileT elements are copied in each row, ignLenY == 0 because no size reduction along Y dim.
		glbPos = tileAddress;	
		moveShareToGlobalEdge(&tile1[0], dev_arr, glbPos, tileX-tileT, 0, tileX, tileY, dep_stride, rowsize, segLengthX, thread);	
		__syncthreads();
*/
		write_tile_lock_for_batch(dev_row_lock, curBatch, thread, yseg, timepiece);
	}

//	write_batch_lock_for_time(timepiece, curBatch);
}


void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void SOR(int n1, int n2, int padd, int *arr, int MAXTRIAL){
	hipSetDevice(0);	
//stride is the longest distance between the element and its dependence along one dimension times
//For example: F(x) = T(x-1) + T(x) + T(x+1), stride = 1
	int stride = 1;
	int dep_stride = stride+1;
	int tileX = 512;
	int tileY = 512;
	int rawElmPerTile = tileX * tileY;
	int tileT = 4;

//PTilesPerTimestamp is the number of parallelgoram tiles can be scheduled at each time stamp
//	int PTilesPerTimestamp = (n1/tileX) * (n2/tileY); 
//ZTilesPerTimestamp is the number of trapezoid tiles (overlaped tiles) needed to calculate the uncovered area at each time stamp.
//	int ZTilesPerTimestamp = (n1/tileX) + (n2/tileY) - 1; 
	int rowsize = 2 * padd + n1; 
	int colsize = 2 * padd + n2;

	volatile int *dev_arr;
	int *lock;
	size_t freeMem, totalMem;
	volatile int *dev_time_lock, *dev_row_lock;	
	
	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes." << endl;
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_arr, tablesize * sizeof(int));
	checkGPUError(err);
	
//	cudaMalloc(&dev_time_lock, n2/tileY * sizeof(int));
	err = hipMemcpy((void*)dev_arr, arr, tablesize*sizeof(int), hipMemcpyHostToDevice);
	checkGPUError(err);
//	cudaMemset((void*)dev_time_lock, 1, n2/tileY * sizeof(int));

	int threadPerBlock = min(1024, rawElmPerTile);
//	int blockPerGrid = PTilesPerTimestamp;
	int blockPerGrid = 1;
	int numStream = 8;
	int warpbatch = threadPerBlock / 32;

//memory structure: stream --> tile --> time --> dependence --> tileX
	int *dev_inter_stream_dependence;
	int stream_dep_offset = tileT * (n1 + dep_stride) * dep_stride;
	int inter_stream_dependence = numStream * stream_dep_offset;
	err = hipMalloc(&dev_inter_stream_dependence, inter_stream_dependence * sizeof(int));
	checkGPUError(err);

	int xseg = n1 / tileX + 1;
	int yseg = n2 / tileY + 1;
	int tseg = (MAXTRIAL + tileT - 1) / tileT;
	int stream_offset = yseg % numStream;
	
	lock = new int[tseg * yseg];
	for (int i = 0; i < tseg; i++){
		int idx = i * yseg;
		lock[idx] = xseg;
		for (int j=1; j<yseg; j++)
			lock[idx+j] = 0;
	}

	err = hipMalloc(&dev_row_lock, tseg * yseg * sizeof(int));
	checkGPUError(err);
	err = hipMemcpy((void*)dev_row_lock, lock, tseg * yseg *sizeof(int), hipMemcpyHostToDevice);
	checkGPUError(err);
	hipStream_t stream[numStream];
	for (int s=0; s<numStream; s++)
		hipStreamCreate(&stream[s]);

//t < MAXTRIAL? or t <= MAXTRIAL	
	for(int t = 0; t < MAXTRIAL; t+= tileT){
//GPU_ZTile() is the kernel function to calculate the update result, unconvered by Parallelgoram tiling.
//These data are calculated with trapezoid tiling, thus they can be launched concurrently.
// ZTile and cudaDeviceSynchronize() will stop theparallelism along the temporal dimension and force
//the beginning of the new t tiles has to wait the completion of the previous t tiles.
//		GPU_ZTile<<<>>>();
//		cudaDeviceSynchronize();		
		for(int curBatch = 0; curBatch < yseg; curBatch++){
//Have to change the stream Index so that the stream for next time tile can start without waiting for the 
//completion of the previous time tile. 
//Example: stream 0, 1, 2 are scheduled to the last three batches in one time tile, since the execution on
//the next time tile also starts from stream 0, this new execution in stream 0 has to wait for the previous
			int logicSMStream = curBatch % numStream;
			int curSMStream = (logicSMStream +  stream_offset * t / tileT) % numStream;
			int curStartAddress = curBatch * tileY * rowsize;
			int rowStartOffset = padd * rowsize + padd;
			int batchStartAddress = rowStartOffset + curStartAddress;
			int nextSMStream = (curSMStream + 1) % numStream;
//			cout << "curBatch: " << curBatch << ", stride: " << stride << ", tileX: " << tileX << ", tileY: " << tileY << ", t: " << t << ", xseg: " << xseg << ", yseg: " << yseg << ", logicStream: " << logicSMStream << ", curStream: " << curSMStream  << endl;	
			GPU_Tile<<<blockPerGrid, threadPerBlock, 0, stream[curSMStream]>>>(dev_arr, curBatch, curStartAddress, tileX, tileY,  padd, stride, rowStartOffset, rowsize, colsize, xseg, yseg, n1, n2, warpbatch, curSMStream, nextSMStream, dev_inter_stream_dependence, inter_stream_dependence, tileT, t, batchStartAddress, dev_row_lock);	
//			GPU_Tile<<<blockPerGrid, threadPerBlock, 0, stream[curSMStream]>>>(stride, tileX, tileY, curBatch, batchStartAddress, dev_row_lock, t, xseg, yseg, tileT);
			checkGPUError( hipGetLastError() );
		}
		//this global synchronization enforces the sequential computation along t dimension.
//		cudaDeviceSynchronize();
	}	
//cudaMemcpy(table, (void*)dev_table, tablesize*sizeof(int), cudaMemcpyDeviceToHost);

	for (int s=0; s<numStream; s++)
		hipStreamDestroy(stream[s]);
	
	hipFree((void*)dev_arr);
	hipFree((void*)dev_row_lock);
	hipFree((void*)dev_inter_stream_dependence);
	delete[] lock;

}

