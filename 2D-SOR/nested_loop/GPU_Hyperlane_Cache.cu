
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

//#define ALL
//#define DEBUG1
//#define DEBUG2
//#define DEBUG3

using namespace std;
__device__ int row = 0;

__device__ void flagRead(int curBatch, volatile int *dev_lock, int thread, int idx, int YoverX, int xseg){
	if (thread == 0){
		int limit = min(idx+YoverX, xseg);
	 	while(dev_lock[curBatch] < limit){
		}
	}
	__syncthreads();
}

__device__ void flagWrite(int curBatch, volatile int *dev_lock, int thread){
	if (thread == 0){
		dev_lock[curBatch+1] += 1;
	}
	__syncthreads();
}

__global__ void GPU(volatile int *dev_table, volatile int *dev_lock, int curBatch, int curStartAddress, int rowtiles, int resX, int tileX, int tileY, int paddsize, int rowStartOffset, int rowsize, int colsize, int xseg, int yseg, int YoverX, int n1, int n2){ 
	//We assume row size n2 is the multiple of 32 and can be completely divided by tileX.
	//on K40, tile size is max to 48K, which is 128*96; on pascal and volta, tile size is max to 64K which is 128*128
	//This code, length of x axis cannot be larger than y axis for each tile.
	//For each row, the first tile and the last tile are computed separately from the other tiles.
	//No padding added, thus the first tile of each row and the first row requires statement check to set dependency to 0 for the edge elements.
	//size of the shared memory is determined by the GPU architecture.
	
#ifdef DEBUG
	if (threadIdx.x == 0){
		printf("This is curBatch: %d, curStartAddress: %d\n", curBatch, curStartAddress);
	}
	__syncthreads();
#endif

	volatile __shared__ int table[12288];

	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int tileStartOffset, lvlStartAddress; 
	int glbStartX;
	int segLengthX = tileX + paddsize;
	int segLengthY = tileY + paddsize;
	int idx;
	int tile = 1;

//***********************************************************************************************************************************
	//processing the first tile of the row
	glbStartX = curStartAddress;
	flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
	
	int highY = tileY;
	int piece = tileY / tileX;

	for (int p = 0; p < piece; p++){
		//first tile is irregular, concurrency is changed from 1 to hightY
		//the x length and y length of the first tile and the last tile are equal.
		tileStartOffset = glbStartX + paddsize * rowsize + paddsize;
		
		//length Y > length X, diagonal first element starts from Y axis instead of X axis for calculating the address.
		int concurrency;
		for (int i=0; i<highY; i++){
			lvlStartAddress = tileStartOffset + i * rowsize;
			concurrency = min(tileX, i);
			if(thread <= concurrency){
				idx = lvlStartAddress - rowsize * thread + thread;
				dev_table[idx] = (dev_table[idx-1] + dev_table[idx-rowsize] 
						+ dev_table[idx] + dev_table[idx+1] + dev_table[idx+rowsize]);
			}
			__syncthreads();
		}
//		__threadfence_system();
#ifdef DEBUG1
#ifdef ALL
		if (thread == 32 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 32){
#endif
			printf("After computation in global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + highY *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
//			}
			printf("\n");
		}
		__syncthreads();
#endif
		glbStartX += tileX;
		highY -= tileX;
//		__syncthreads();
	}
	//update the tile beginning pos for the next tile.
//	glbStartX += (tileY + 1);
	glbStartX += 1;
	flagWrite(curBatch, dev_lock, thread);
//***********************************************************************************************************************************
	//hyperlane tiles
	for (tile = 2; tile < xseg; tile++){
		tileStartOffset = glbStartX + paddsize * rowsize + paddsize - 1;
		flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
		
		lvlStartAddress = tileStartOffset;
		for (int i=0; i<tileX; i++){
//this is expensive especially when tileX is large. However, if we put if statement outside the loop, we face syncthreads issue.
//New feature warp level syncronize or thread group syncronize may solve the problem.
			if (thread < tileY){
				idx = lvlStartAddress + thread * rowsize - thread;
				dev_table[idx] = (dev_table[idx-1] + dev_table[idx-rowsize] 
						+ dev_table[idx] + dev_table[idx+1] + dev_table[idx+rowsize]);
			}
			lvlStartAddress += 1;
			__syncthreads();
		}
//		__threadfence_system();

#ifdef DEBUG2
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation, global memory. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j - 1]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-tileX) *rowsize+j - (segLengthY-tileX-1)]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j - (segLengthY-1)]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j - i * (segLengthY-1)]);
				//}
				//printf("\n");
		//	}
			printf("\n");
		}
		__syncthreads();
#endif

		//update the tile beginning pos for the next tile.
		glbStartX += tileX;
		flagWrite(curBatch, dev_lock, thread);
	}

//************************************************************************************************************************************
	//the last tile, which is a half of the rectangular
	flagRead(curBatch, dev_lock, thread, xseg, YoverX, xseg);
	glbStartX = curStartAddress + rowsize - paddsize - tileY - paddsize;
	
	piece = tileY / tileX;
	highY = tileX;
	
	for (int p=0; p<piece; p++){
		//last tile is irregular, concurrency is changed from hightY-1 to 1
		//the x length and y length of the first tile and the last tile are equal.
		int concurrency;
		tileStartOffset = glbStartX + rowsize * (segLengthY - highY);
		for (int i=highY; i>0; i--){
			lvlStartAddress = tileStartOffset + segLengthX - 1;
			concurrency = min(tileX, i);	
			if(thread < concurrency){
				idx = lvlStartAddress + rowsize * thread - thread;
				dev_table[idx] = (dev_table[idx-1] + dev_table[idx-rowsize] 
						+ dev_table[idx] + dev_table[idx+1] + dev_table[idx+rowsize]);
			}
			tileStartOffset += rowsize;
			__syncthreads();
		}
//		__threadfence_system();	

#ifdef DEBUG3
#ifdef ALL
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation. global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + 2 * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
			//}
			printf("\n");
		}
		__syncthreads();
#endif
		glbStartX += tileX;
		highY += tileX;
	}
	
	flagWrite(curBatch, dev_lock, thread);
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void SOR(int n1, int n2, int *table){
	hipSetDevice(0);
	hipDeviceProp_t gpuinfo;
	hipGetDeviceProperties(&gpuinfo, 0);
	int paddsize = 1;
	//tileY must be larger than tileX
	int tileX = 128;
	int tileY = 1024;
	int rowsize = paddsize * 2 + n1;
	int colsize = paddsize * 2 + n2;

	volatile int *dev_table, *dev_lock;
	int *lock;
	size_t freeMem, totalMem;
	
	
	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
#ifdef DEBUG
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
#endif	
	hipError_t err = hipMalloc(&dev_table, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMemcpy((void*)dev_table, table, tablesize*sizeof(int), hipMemcpyHostToDevice);

	int threadPerBlock = max(tileY, tileX);
	int blockPerGrid = 1;
	int numStream = gpuinfo.multiProcessorCount;

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	//For hyperlane tiles, if tileX!=tileY, the X length of the first tile and the last tile are equal to tileY.
//	int xseg = (n1+tileX-1) / tileX;
	int xseg = ((n1-tileY) + tileX - 1) / tileX + 2;
	int yseg = (n2+tileY-1) / tileY;

	lock = new int[yseg+1];
	lock[0] = xseg+1;
	for (int i=1; i<yseg+1; i++)
		lock[i] = 0;
	hipMalloc(&dev_lock, (yseg+1) * sizeof(int));	
//	cudaMemset((void*)dev_lock, 0, (yseg + 1) * sizeof(int));
//	cudaMemset((void*)dev_lock, xseg+1, sizeof(int));
	hipMemcpy((void*)dev_lock, lock, (yseg+1)*sizeof(int), hipMemcpyHostToDevice);
	hipStream_t stream[numStream];
	for (int s=0; s<numStream; s++)
		hipStreamCreate(&stream[s]);
	
	//instead of calling kernels along anti-diagonals, we now schedule kernels for each row.
	//We assume X axis is completly divided by tileX
	for(int curBatch = 0; curBatch < yseg; curBatch++){
		int curSMStream = curBatch % numStream;
		//int resY = n1 - curBatch * tileY;
		int resX = (n2 - tileY) % tileX;
		int curStartAddress = curBatch * tileY * rowsize;
		int rowStartOffset = paddsize * rowsize + paddsize;
		int rowtiles = xseg + 1;
		GPU<<<blockPerGrid, threadPerBlock, 0, stream[curSMStream]>>>(dev_table, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY, paddsize, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
//		GPU<<<blockPerGrid, threadPerBlock>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
//		cudaDeviceSynchronize();
	}
	hipDeviceSynchronize();
	//cudaMemcpy(table, (void*)dev_table, tablesize*sizeof(int), cudaMemcpyDeviceToHost);

#ifdef DEBUG
	//display table
	cout << "grid size: " << blockPerGrid << ", block size: " << threadPerBlock << ", full table: " << endl;
	for (int i=0; i<colsize; i++){
		for (int j=0; j<rowsize; j++){
			cout << table[i * rowsize + j] << " ";
		}
		cout << endl;
	}
	
#endif
	for (int s=0; s<numStream; s++)
		hipStreamDestroy(stream[s]);
	
	hipFree((void*)dev_table);
	hipFree((void*)dev_lock);
	delete[] lock;
}

