
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

//#define ALL
//#define DEBUG1
//#define DEBUG2
//#define DEBUG3

using namespace std;
__device__ int row = 0;

__device__ void flagRead(int curBatch, volatile int *dev_lock, int thread, int idx, int YoverX, int xseg){
	if (thread == 0){
		int limit = min(idx+YoverX, xseg);
/*
#ifdef DEBUG		
		printf("curBatch: %d, tile: %d, limit: %d, dev_lock[curBatch]: %d\n", curBatch, idx, limit, dev_lock[curBatch]);
#endif
*/
	 	while(dev_lock[curBatch] < limit){
		}
/*
#ifdef DEBUG
		printf("curBatch: %d, tile: %d, is permit to proceed, dev_lock[curBatch]: %d\n", curBatch, idx, dev_lock[curBatch]);
#endif
*/
	}
	__syncthreads();
}

__device__ void flagWrite(int curBatch, volatile int *dev_lock, int thread){
	if (thread == 0){
		dev_lock[curBatch+1] += 1;
	}
	__syncthreads();
}

__global__ void GPU(volatile int *dev_table, int *dev_arr1, int *dev_arr2, volatile int *dev_lock, int curBatch, int curStartAddress, int rowtiles, int resX, int tileX, int tileY, int paddX, int paddY, int rowStartOffset, int rowsize, int colsize, int xseg, int yseg, int YoverX, int n1, int n2){ 
	//We assume row size n2 is the multiple of 32 and can be completely divided by tileX.
	//on K40, tile size is max to 48K, which is 128*96; on pascal and volta, tile size is max to 64K which is 128*128
	//This code, length of x axis cannot be larger than y axis for each tile.
	//For each row, the first tile and the last tile are computed separately from the other tiles.
	//No padding added, thus the first tile of each row and the first row requires statement check to set dependency to 0 for the edge elements.
	//size of the shared memory is determined by the GPU architecture.
	
#ifdef DEBUG
	if (threadIdx.x == 0){
		printf("This is curBatch: %d, curStartAddress: %d\n", curBatch, curStartAddress);
	}
	__syncthreads();
#endif

	volatile __shared__ int table[12288];

	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int tileStartOffset, lvlStartAddress; 
	int glbStartX;
	int segLengthX = tileX + paddX;
	int segLengthY = tileY + paddY;
	int arrX = 0;
	int arrY = curBatch * tileY;
	int idxx, idxy, idx;
	int tile = 1;

//***********************************************************************************************************************************
	//processing the first tile of the row
	glbStartX = curStartAddress;
	flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
	
	int highY = tileY;
	int piece = tileY / tileX;

	for (int p = 0; p < piece; p++){
		//first tile is irregular, concurrency is changed from 1 to hightY
		//the x length and y length of the first tile and the last tile are equal.
		tileStartOffset = glbStartX + paddY * rowsize + paddX;
		
		//length Y > length X, diagonal first element starts from Y axis instead of X axis for calculating the address.
		int concurrency;
		for (int i=0; i<highY; i++){
			lvlStartAddress = tileStartOffset + i * rowsize;
			concurrency = min(tileX, i);
			if(thread <= concurrency){
				idx = lvlStartAddress - rowsize * thread + thread;
				idxx = arrX + thread;
				idxy = arrY + i - thread;
				dev_table[idx] = max(dev_table[idx-1], dev_table[idx-rowsize]);
				if(dev_arr1[idxx] == dev_arr2[idxy]){
					dev_table[idx] = dev_table[idx-rowsize-1] + 1;	
				}	
	//			printf("curBatch: %d, tile: %d, thread: %d, idx: %d, idxx: %d, x[idxx]: %d, idxy: %d, y[idxy]: %d, table[idx]: %d\n", curBatch, tile, thread, idx, idxx, dev_arr1[idxx], idxy, dev_arr2[idxy], table[idx]);						
			}
			__syncthreads();
		}
		__threadfence_system();
#ifdef DEBUG1
#ifdef ALL
		if (thread == 32 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 32){
#endif
			printf("After computation in global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + highY *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
//			}
			printf("\n");
		}
		__syncthreads();
#endif
		arrX += tileX;		
		glbStartX += tileX;
		highY -= tileX;
		__syncthreads();
	}
	//update the tile beginning pos for the next tile.
//	glbStartX += (tileY + 1);
	glbStartX += 1;
	arrX = tileY;
	flagWrite(curBatch, dev_lock, thread);
//***********************************************************************************************************************************
	//hyperlane tiles
	for (tile = 2; tile < xseg; tile++){
		tileStartOffset = glbStartX + paddY * rowsize + paddX - 1;
		flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
		
		lvlStartAddress = tileStartOffset;
		for (int i=0; i<tileX; i++){
//this is expensive especially when tileX is large. However, if we put if statement outside the loop, we face syncthreads issue.
//New feature warp level syncronize or thread group syncronize may solve the problem.
			if (thread < tileY){
				idx = lvlStartAddress + thread * rowsize - thread;
				idxx = arrX + (i - thread);
				idxy = arrY + thread;
				dev_table[idx] = max(dev_table[idx-1], dev_table[idx-rowsize]);
				if(dev_arr1[idxx] == dev_arr2[idxy]){
					dev_table[idx] = dev_table[idx-rowsize-1] + 1;
				}	
			}
			lvlStartAddress += 1;
			__syncthreads();
		}
		__threadfence_system();

#ifdef DEBUG2
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation, global memory. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j - 1]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-tileX) *rowsize+j - (segLengthY-tileX-1)]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j - (segLengthY-1)]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j - i * (segLengthY-1)]);
				//}
				//printf("\n");
		//	}
			printf("\n");
		}
		__syncthreads();
#endif

		//update the tile beginning pos for the next tile.
		glbStartX += tileX;
		arrX+=tileX;
		flagWrite(curBatch, dev_lock, thread);
	}

//************************************************************************************************************************************
	//the last tile, which is a half of the rectangular
	flagRead(curBatch, dev_lock, thread, xseg, YoverX, xseg);
	glbStartX = curStartAddress + rowsize - tileY - paddX;
	
	piece = tileY / tileX;
	highY = tileX;
	
	for (int p=0; p<piece; p++){
		//last tile is irregular, concurrency is changed from hightY-1 to 1
		//the x length and y length of the first tile and the last tile are equal.
		int concurrency;
		tileStartOffset = glbStartX + rowsize * (segLengthY - highY);
		for (int i=highY; i>0; i--){
			lvlStartAddress = tileStartOffset + segLengthX - 1;
			concurrency = min(tileX, i);	
			if(thread < concurrency){
				idx = lvlStartAddress + rowsize * thread - thread;
				idxx = (n1 - tileY + highY - 1) - thread;
				idxy = (arrY + tileY - i) + thread;
				dev_table[idx] = max(dev_table[idx-1], dev_table[idx-rowsize]);
				if(dev_arr1[idxx] == dev_arr2[idxy]){
					dev_table[idx] = dev_table[idx-rowsize-1] + 1;	
				}							
			}
			tileStartOffset += rowsize;
			__syncthreads();
		}
		__threadfence_system();	

#ifdef DEBUG3
#ifdef ALL
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation. global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + 2 * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
			//}
			printf("\n");
		}
		__syncthreads();
#endif
		glbStartX += tileX;
		highY += tileX;
	}
	
	flagWrite(curBatch, dev_lock, thread);
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

int LCS(int n1, int n2, int *arr1, int *arr2, int paddX, int paddY, int *table){
	int lcslength;

	//tileY must be larger than tileX
	int tileX = 256;
	int tileY = 256;
	int rowsize = paddX + n2;
	int colsize = paddY + n1;

	int *dev_arr1, *dev_arr2;
	volatile int *dev_table, *dev_lock;
	int *lock;
	size_t freeMem, totalMem;
	
	
	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_table, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMalloc(&dev_arr1, n1*sizeof(int));
	hipMalloc(&dev_arr2, n2*sizeof(int));

	hipMemset((void*)dev_table, 0, tablesize * sizeof(int));
	hipMemcpy(dev_arr1, arr1, n1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr2, arr2, n2*sizeof(int), hipMemcpyHostToDevice);

	int threadPerBlock = max(tileY, tileX);
	int blockPerGrid = 1;
	int numStream = 15;

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	//For hyperlane tiles, if tileX!=tileY, the X length of the first tile and the last tile are equal to tileY.
//	int xseg = (n1+tileX-1) / tileX;
	int xseg = ((n1-tileY) + tileX - 1) / tileX + 2;
	int yseg = (n2+tileY-1) / tileY;

	lock = new int[yseg+1];
	lock[0] = xseg+1;
	for (int i=1; i<yseg+1; i++)
		lock[i] = 0;
	hipMalloc(&dev_lock, (yseg+1) * sizeof(int));	
//	cudaMemset((void*)dev_lock, 0, (yseg + 1) * sizeof(int));
//	cudaMemset((void*)dev_lock, xseg+1, sizeof(int));
	hipMemcpy((void*)dev_lock, lock, (yseg+1)*sizeof(int), hipMemcpyHostToDevice);
	hipStream_t stream[numStream];
	for (int s=0; s<numStream; s++)
		hipStreamCreate(&stream[s]);
	
	//instead of calling kernels along anti-diagonals, we now schedule kernels for each row.
	//We assume X axis is completly divided by tileX
	for(int curBatch = 0; curBatch < yseg; curBatch++){
		int curSMStream = curBatch % numStream;
		//int resY = n1 - curBatch * tileY;
		int resX = (n2 - tileY) % tileX;
		int curStartAddress = curBatch * tileY * rowsize;
		int rowStartOffset = paddY * rowsize + paddX;
		int rowtiles = xseg + 1;
//		cout << endl << "curBatch: " << curBatch << ", yseg: " << yseg << endl;	
		GPU<<<blockPerGrid, threadPerBlock, 0, stream[curSMStream]>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
//		GPU<<<blockPerGrid, threadPerBlock>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
//		cudaDeviceSynchronize();
	}
	hipMemcpy(&lcslength, (void*)&dev_table[tablesize-1], sizeof(int), hipMemcpyDeviceToHost);
	//cudaMemcpy(table, (void*)dev_table, tablesize*sizeof(int), cudaMemcpyDeviceToHost);

#ifdef DEBUG
	//display table
	cout << "grid size: " << blockPerGrid << ", block size: " << threadPerBlock << ", full table: " << endl;
	for (int i=0; i<colsize; i++){
		for (int j=0; j<rowsize; j++){
			cout << table[i * rowsize + j] << " ";
		}
		cout << endl;
	}
	
#endif
	for (int s=0; s<numStream; s++)
		hipStreamDestroy(stream[s]);
	
	hipFree(dev_arr1);
	hipFree(dev_arr2);
	hipFree((void*)dev_table);
	hipFree((void*)dev_lock);
	delete[] lock;

	return lcslength;
}

