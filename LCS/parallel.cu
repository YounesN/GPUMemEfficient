
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

typedef unsigned long long int UINT;

using namespace std;

__global__ void GPU(int *dev_table, int *dev_arr1, int *dev_arr2, int startIdx, int curjobs, const int rowsize, int startx, int starty){
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (thread < curjobs){
		int idx = startIdx + (thread * rowsize - thread);
		int x = startx - thread;
		int y = starty + thread;
		
		*(dev_table + idx) = max(*(dev_table+idx-1), *(dev_table+idx-rowsize));
		
		if (dev_arr1[x] == dev_arr2[y]){
//			printf("if equal, before, idx: %d, dev_table[idx]: %d; pre: %d, dev_table[idx-rowsize-1]: %d \n", idx, dev_table[idx], idx-rowsize-1, dev_table[idx-rowsize-1]);
			*(dev_table+idx) = *(dev_table+ (idx - rowsize - 1) ) + 1;
//			printf("after, dev_table[idx]: %d\n", dev_table[idx]);
		}		
//		printf("thread: %d, idx: %d, x: %d, y: %d, arr1[x]: %d, arr2[y]: %d \n", thread, idx, x, y, dev_arr1[x], dev_arr2[y]);
	}
//	__threadfence();
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

int LCS(int n1, int n2, int *arr1, int *arr2){
	int lcslength;
	int poolsize = 32;
	int rowsize = poolsize + n2;
	int colsize = poolsize + n1;

	int *dev_table, *dev_arr1, *dev_arr2;

	//int *table;

	//table = new int[(n1+poolsize) * rowsize];

	size_t freeMem, totalMem;

	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_table, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMalloc(&dev_arr1, n1*sizeof(int));
	hipMalloc(&dev_arr2, n2*sizeof(int));

	hipMemset(dev_table, 0, tablesize * sizeof(int));
	hipMemcpy(dev_arr1, arr1, n1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr2, arr2, n2*sizeof(int), hipMemcpyHostToDevice);

	int maxthreads = min(n1, n2);
	int maxlevel = n1 + n2 - 1;	
	int curlevel = 1;
	int curjobs = 1;
	int startx, starty;
	int threadPerBlock = 96, blockPerGrid;

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	//suppose n2 is the row size and the longer array
	while(curlevel <= maxlevel){
//		cout << "level: " << curlevel << endl;
		int startIdx;
		if (curlevel <= n2){
			startIdx = curlevel - 1;
			curjobs = curlevel;
			startx = startIdx;
			starty = 0;
		}
		else{
			startIdx = n2 - 1 + rowsize * (curlevel - n2);
			curjobs = 2 * n2 - curlevel;
			startx = n2 - 1;
			starty = curlevel - n2;
		}

		int numthreads = (curjobs + 31) / 32;
		numthreads *= 32;
	
		blockPerGrid = (numthreads + threadPerBlock - 1) / threadPerBlock;

		GPU<<<blockPerGrid, threadPerBlock>>>(&dev_table[poolsize*rowsize+poolsize], dev_arr1, dev_arr2, startIdx, curjobs, rowsize, startx, starty);		
		
		hipDeviceSynchronize();		

		curlevel++;
	}

	hipMemcpy(&lcslength, &dev_table[tablesize-1], sizeof(int), hipMemcpyDeviceToHost);
	//cudaMemcpy(table, dev_table, (n1+poolsize)*rowsize*sizeof(int), cudaMemcpyDeviceToHost);
/*
	//display table
	cout << "full table: " << endl;
	for (int i=0; i<n1+poolsize; i++){
		for (int j=0; j<n2+poolsize; j++){
			cout << table[i * rowsize + j] << " ";
		}
		cout << endl;
	}
*/	

	hipFree(dev_arr1);
	hipFree(dev_arr2);
	hipFree(dev_table);

//	delete[] table;

	return lcslength;
}

