
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

#define ALL
#define DEBUG

using namespace std;
__device__ int row = 0;

__device__ void moveToShare(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY, int paddX){
	//potential bank conflict for accessing the data of each anti-diagonal
	//to avoid bank conflict, have to adjust the memory layout by introducing dummy elements.
	//padding elements can be used as the dummy elements, but the number of padding of each dimension has to be an odd number.
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
//	if (thread < paddX){
		for (int i=0; i<segLengthY; i++){
//			printf("thread: %d, segLengthX: %d, pos: %d, idx: %d\n", thread, segLengthX, pos, idx);			
			
			table[idx] = dev_table[pos];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
}

__device__ void moveToGlobal(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY){
	int pos = tileAddress + thread;
	int idx = thread;
	//If y dimension cannot be completely divided by tileY, this code causes errors.
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			dev_table[pos] = table[idx];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
}

__device__ void moveToShareRec(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY){
	//This function is designed for the first and the last tiles, which can be treate as rectangular but not hyperlane.
	//Rectangular tile does not have bank conflict issue.
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			table[idx] = dev_table[pos];
			pos += rowsize;
			idx += segLengthX;
		}	
	}
}

__device__ void moveToGlobalRec(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY){
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			dev_table[pos] = table[idx];
			pos += rowsize;
			idx += segLengthX;
		}
	}	
}

__device__ void moveToShareLast(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY){
	//This function is designed for the last tiles, which can be treate as rectangular but not hyperlane.
	//Rectangular tile does not have bank conflict issue.
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < hightY){
		for (int i=0; i<hightY; i++){
			table[idx] = dev_table[pos];
			pos += rowsize;
			idx += hightY;
		}	
	}
}

__device__ void moveToGlobalLast(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY){
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < hightY){
		for (int i=0; i<hightY; i++){
			dev_table[pos] = table[idx];
			pos += rowsize;
			idx += hightY;
		}
	}
}

__device__ void flagRead(int curBatch, volatile int *dev_lock, int thread, int idx, int YoverX, int xseg){
	if (thread == 0){
		int limit = min(idx+YoverX, xseg);
/*
#ifdef DEBUG		
		printf("curBatch: %d, tile: %d, limit: %d, dev_lock[curBatch]: %d\n", curBatch, idx, limit, dev_lock[curBatch]);
#endif
*/
	 	while(dev_lock[curBatch] < limit){
		}
/*
#ifdef DEBUG
		printf("curBatch: %d, tile: %d, is permit to proceed, dev_lock[curBatch]: %d\n", curBatch, idx, dev_lock[curBatch]);
#endif
*/
	}
	__syncthreads();
}

__device__ void flagWrite(int curBatch, volatile int *dev_lock, int thread){
	if (thread == 0){
		dev_lock[curBatch+1] += 1;
	}
	__syncthreads();
}

__global__ void GPU(volatile int *dev_table, int *dev_arr1, int *dev_arr2, volatile int *dev_lock, int curBatch, int curStartAddress, int rowtiles, int hightY, int resX, int tileX, int tileY, int paddX, int paddY, int rowStartOffset, int rowsize, int colsize, int xseg, int yseg, int YoverX, int n1, int n2){ 
	//We assume row size n2 is the multiple of 32 and can be completely divided by tileX.
	//on K40, tile size is max to 48K, which is 128*96; on pascal and volta, tile size is max to 64K which is 128*128
	//This code, length of x axis cannot be larger than y axis for each tile.
	//For each row, the first tile and the last tile are computed separately from the other tiles.
	//No padding added, thus the first tile of each row and the first row requires statement check to set dependency to 0 for the edge elements.
	//size of the shared memory is determined by the GPU architecture.
	
	volatile __shared__ int table[12288];

	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int tileStartOffset, lvlStartAddress; 
	int glbStartX;
	int segLengthX = tileX + paddX;
	int segLengthY = tileY + paddY;
	int arrX = 0;
	int arrY = curBatch * tileY;
	int idxx, idxy, idx;
	int tile = 1;

//***********************************************************************************************************************************
	//processing the first tile of the row
	glbStartX = curStartAddress;
	flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
	

#ifdef DEBUG
#ifdef ALL
		if (thread == 0 && curBatch == row ){	
#endif
#ifndef ALL
		if (thread == 0){
#endif
			printf("Before move data to share memory. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
#endif		

	moveToShareRec(&table[0], dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);				
	__syncthreads();
	__threadfence_block();
#ifdef DEBUG
#ifdef ALL	
	if (thread == 32 && curBatch == row){	
#endif
#ifndef ALL
	if (thread == 32){
#endif	
		printf("Before computation, share memory. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
		//for (int i=0; i<segLengthY; i++){

			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
		printf("\n");
	}
#endif
	//first tile is irregular, concurrency is changed from 1 to hightY
	//the x length and y length of the first tile and the last tile are equal.
	tileStartOffset = paddY * segLengthX + paddX;
	for (int i=0; i<hightY; i++){
		lvlStartAddress = tileStartOffset + i;
		
		if(thread <= i){
			idx = lvlStartAddress + segLengthX * thread - thread;
			idxx = arrX + (i - thread);
			idxy = arrY + thread;
			table[idx] = max(table[idx-1], table[idx-segLengthX]);
			if(dev_arr1[idxx] == dev_arr2[idxy]){
				table[idx] = table[idx-segLengthX-1] + 1;	
			}	
//			printf("curBatch: %d, tile: %d, thread: %d, idx: %d, idxx: %d, x[idxx]: %d, idxy: %d, y[idxy]: %d, table[idx]: %d\n", curBatch, tile, thread, idx, idxx, dev_arr1[idxx], idxy, dev_arr2[idxy], table[idx]);						
		}
		__syncthreads();
	}

#ifdef DEBUG
#ifdef ALL	
	if (thread == 32 && curBatch==row){	
#endif
#ifndef ALL
	if (thread == 32){
#endif
		printf("After computation, in shared memory\n");
		//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
		printf("\n");
	}
#endif

	moveToGlobalRec(&table[0], dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);				
	__threadfence_block();
	__syncthreads();


#ifdef DEBUG
#ifdef ALL
	if (thread == 32 && curBatch == row){
#endif
#ifndef ALL
	if (thread == 32){
#endif
		printf("After computation in global memory. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
		//for(int i=0; i<segLengthY; i++){
			for(int j=0; j<segLengthX; j++){	
				printf("%d ", dev_table[glbStartX + rowsize+j]);
			}
			printf("\n");
			for(int j=0; j<segLengthX; j++){	
				printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j]);
			}
			printf("\n");
			//for(int j=0; j<segLengthX; j++){	
			//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
			//}
			//printf("\n");
//		}
		printf("\n");
	}
#endif

	//update the tile beginning pos for the next tile.
	glbStartX += (tileY + 1);
	arrX = tileY;
	flagWrite(curBatch, dev_lock, thread);
//***********************************************************************************************************************************
	//hyperlane tiles
	tileStartOffset = paddY * segLengthX + paddX;
	for (tile = 2; tile < xseg; tile++){
		flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
	
//		printf("curBatch: %d, tile: %d, thread: %d is permit to read data from global.\n", curBatch, tile, thread);

#ifdef DEBUG
#ifdef ALL		
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("Before computation global memory. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j - 1]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j - (segLengthY-1)]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j - i * (segLengthY-1)]);
				//}
				//printf("\n");
//			}
			printf("\n");
		}
#endif	

#ifdef DEBUG
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3 ){	
#endif
#ifndef ALL
		if (thread == 0){
#endif
			printf("Before move data to share memory. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
#endif		
		moveToShare(&table[0], dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY, paddX);
		__syncthreads();
		__threadfence_system();
/*
#ifdef DEBUG
#ifdef ALL		
		if (thread == 0 && curBatch == row && tile <=3){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("Before computation global. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + i*rowsize+j - i]);
				}
				printf("\n");
			}
			printf("\n");
		}
#endif
*/	

#ifdef DEBUG
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3 ){	
#endif
#ifndef ALL
		if (thread == 0){
#endif
			printf("Before computation share. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
#endif		

		lvlStartAddress = tileStartOffset;
		for (int i=0; i<tileX; i++){
//this is expensive especially when tileX is large. However, if we put if statement outside the loop, we face syncthreads issue.
//New feature warp level syncronize or thread group syncronize may solve the problem.
			if (thread < tileY){
				idx = lvlStartAddress + thread * segLengthX;
				idxx = arrX + (i - thread);
				idxy = arrY + thread;
				table[idx] = max(table[idx-1], table[idx-segLengthX-1]);
				if(dev_arr1[idxx] == dev_arr2[idxy]){
					table[idx] = table[idx-segLengthX-2] + 1;
				}	
				lvlStartAddress += 1;
			}
			__syncthreads();
		}

#ifdef DEBUG
#ifdef ALL
	if (thread == 0 && curBatch == row && tile<=3){	
#endif
#ifndef ALL
	if (thread == 0){
#endif
		printf("After computation, in shared memory.\n");
		//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
		printf("\n");
	}
#endif

		//need modification, only copy the new updated elements back to the global memory. Also modify moveToGlobalRec
		moveToGlobal(&table[0], dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);
//		moveToGlobal(&table[paddX], dev_table, glbStartX + paddX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);
		
		__threadfence_system();
		__syncthreads();

#ifdef DEBUG
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation, global memory. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j - 1]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j - (segLengthY-1)]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j - i * (segLengthY-1)]);
				//}
				//printf("\n");
		//	}
			printf("\n");
		}
#endif

		//update the tile beginning pos for the next tile.
		glbStartX += tileX;
		arrX+=tileX;
		flagWrite(curBatch, dev_lock, thread);
	}

//************************************************************************************************************************************
	//the last tile, which is a half of the rectangular
	flagRead(curBatch, dev_lock, thread, xseg, YoverX, xseg);

	glbStartX = curStartAddress + rowsize * paddY + rowsize - tileX;
#ifdef DEBUG
#ifdef ALL	
	if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
	if (thread == 0){
#endif
		printf("Before computation global memory. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
		//for(int i=0; i<segLengthY; i++){
			for(int j=0; j<segLengthX; j++){	
				printf("%d ", dev_table[glbStartX + rowsize+j]);
			}
			printf("\n");
			for(int j=0; j<segLengthX; j++){	
				printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j]);
			}
			printf("\n");
			//for(int j=0; j<segLengthX; j++){	
			//	printf("%d ", dev_table[glbStartX + i* (segLengthY-1) *rowsize+j]);
			//}
			//printf("\n");
		//}
		printf("\n");
	}
#endif
	
	moveToShareLast(&table[0], dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);				
	__syncthreads();
	__threadfence_block();

#ifdef DEBUG
#ifdef ALL
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif
			printf("last tile share memory, glbStartX: %d, rowsize: %d, hightY: %d\n", glbStartX, rowsize, hightY);	
		//	for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
#endif

	//last tile is irregular, concurrency is changed from hightY-1 to 1
	//the x length and y length of the first tile and the last tile are equal.
	tileStartOffset = hightY;
	for (int i=hightY-1; i>0; i--){
		lvlStartAddress = tileStartOffset + hightY - 1;
		
		if(thread < i){
			idx = lvlStartAddress + hightY * thread - thread;
			idxx = (n1 - 1) - thread;
			idxy = (arrY + 1) + thread;
			table[idx] = max(table[idx-1], table[idx-hightY]);
			if(dev_arr1[idxx] == dev_arr2[idxy]){
				table[idx] = table[idx-hightY-1] + 1;	
			}							
#ifdef DEBUG
				if(curBatch == row){
	//				printf("curBatch: %d, thread: %d, idx: %d, idxx: %d, arr1[idxx]: %d, idxy: %d, arr2[idxy]: %d, table[idx]: %d\n", curBatch, thread, idx, idxx, dev_arr1[idxx], idxy, dev_arr2[idxy], table[idx]);						
				}
#endif
		}
		arrY += 1;
		tileStartOffset += hightY;
		__syncthreads();
	}
	
	moveToGlobalLast(&table[0], dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);				
	__syncthreads();	

#ifdef DEBUG
#ifdef ALL	
	if (thread == 0 && curBatch == row){	
#endif
#ifndef ALL
	if (thread == 0){
#endif	
		printf("After computation, in shared memory\n");
		//for (int i=0; i<hightY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
		printf("\n");
	}
#endif


#ifdef DEBUG
#ifdef ALL
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation. global memory. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
			//}
			printf("\n");
		}
#endif

	flagWrite(curBatch, dev_lock, thread);
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

int LCS(int n1, int n2, int *arr1, int *arr2){
	int lcslength;
	int paddX = 3, paddY = 1;

	//tileY must be larger than tileX
	int tileX = 64;
	int tileY = 128;
	int rowsize = paddX + n2;
	int colsize = paddY + n1;

	int *dev_arr1, *dev_arr2;
	volatile int *dev_table, *dev_lock;
	int *lock;
	size_t freeMem, totalMem;
#ifdef DEBUG
	int *table = new int[colsize * rowsize];
#endif
	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_table, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMalloc(&dev_arr1, n1*sizeof(int));
	hipMalloc(&dev_arr2, n2*sizeof(int));

	hipMemset((void*)dev_table, 0, tablesize * sizeof(int));
	hipMemcpy(dev_arr1, arr1, n1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr2, arr2, n2*sizeof(int), hipMemcpyHostToDevice);

	int threadPerBlock = max(tileY + 32, tileX + 32);
	int blockPerGrid = 1;
	int numStream = 15;

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	//For hyperlane tiles, if tileX!=tileY, the X length of the first tile and the last tile are equal to tileY.
//	int xseg = (n1+tileX-1) / tileX;
	int xseg = ((n1-tileY) + tileX - 1) / tileX + 2;
	int yseg = (n2+tileY-1) / tileY;

	lock = new int[yseg+1];
	lock[0] = xseg+1;
	for (int i=1; i<yseg+1; i++)
		lock[i] = 0;
	hipMalloc(&dev_lock, (yseg+1) * sizeof(int));	
//	cudaMemset((void*)dev_lock, 0, (yseg + 1) * sizeof(int));
//	cudaMemset((void*)dev_lock, xseg+1, sizeof(int));
	hipMemcpy((void*)dev_lock, lock, (yseg+1)*sizeof(int), hipMemcpyHostToDevice);
	hipStream_t stream[numStream];
	for (int s=0; s<numStream; s++)
		hipStreamCreate(&stream[s]);
	
	//instead of calling kernels along anti-diagonals, we now schedule kernels for each row.
	//We assume X axis is completly divided by tileX
	for(int curBatch = 0; curBatch < yseg; curBatch++){
		int curSMStream = curBatch % yseg;
		int resY = n1 - curBatch * tileY;
		int hightY = min(tileY, resY);
		int resX = (n2 - tileY) % tileX;
		int curStartAddress = curBatch * tileY * rowsize;
		int rowStartOffset = paddY * rowsize + paddX;
		int rowtiles = xseg + 1;
	
//		GPU<<<blockPerGrid, threadPerBlock, 0, stream[curSMStream]>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, hightY, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
		GPU<<<blockPerGrid, threadPerBlock>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, hightY, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
		hipDeviceSynchronize();
	}
	hipDeviceSynchronize();	
	hipMemcpy(&lcslength, (void*)&dev_table[tablesize-1], sizeof(int), hipMemcpyDeviceToHost);
#ifdef DEBUG
	hipMemcpy(table, (void*)dev_table, tablesize*sizeof(int), hipMemcpyDeviceToHost);
/*
	//display table
	cout << "grid size: " << blockPerGrid << ", block size: " << threadPerBlock << ", full table: " << endl;
	for (int i=0; i<colsize; i++){
		for (int j=0; j<rowsize; j++){
			cout << table[i * rowsize + j] << " ";
		}
		cout << endl;
	}
*/	
#endif
	for (int s=0; s<numStream; s++)
		hipStreamDestroy(stream[s]);
	
	hipFree(dev_arr1);
	hipFree(dev_arr2);
	hipFree((void*)dev_table);
	hipFree((void*)dev_lock);
#ifdef DEBUG	
	delete[] table;
#endif	
	delete[] lock;

	return lcslength;
}

