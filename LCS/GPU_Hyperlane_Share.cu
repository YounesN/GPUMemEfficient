
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

typedef unsigned long long int UINT;

using namespace std;
__device__ int row = 1;

__device__ void moveToShare(int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY){
	//potential bank conflict for accessing the data of each anti-diagonal
	//to avoid bank conflict, have to adjust the memory layout by introducing dummy elements.
	//padding elements can be used as the dummy elements, but the number of padding of each dimension has to be an odd number.
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
//printf("thread: %d, pos: %d, idx: %d\n", thread, pos, idx);			
			
			table[idx] = dev_table[pos];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
}

__device__ void moveToGlobal(int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY){
	int pos = tileAddress + thread;
	int idx = thread;
	//If y dimension cannot be completely divided by tileY, this code causes errors.
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){

//printf("thread: %d, segLengthX: %d, pos: %d, idx: %d\n", thread, segLengthX, pos, idx);			
			dev_table[pos] = table[idx];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
}

__device__ void moveToShareRec(int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY){
	//This function is designed for the first and the last tiles, which can be treate as rectangular but not hyperlane.
	//Rectangular tile does not have bank conflict issue.
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			table[idx] = dev_table[pos];
			pos += rowsize;
			idx += segLengthX;
		}	
	}
}

__device__ void moveToGlobalRec(int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY){
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			dev_table[pos] = table[idx];
			pos += rowsize;
			idx += segLengthX;
		}
	}	
}

__device__ void moveToShareLast(int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY){
	//This function is designed for the last tiles, which can be treate as rectangular but not hyperlane.
	//Rectangular tile does not have bank conflict issue.
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < hightY){
		for (int i=0; i<hightY; i++){
			table[idx] = dev_table[pos];
			pos += rowsize;
			idx += hightY;
		}	
	}
}

__device__ void moveToGlobalLast(int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int hightY, int segLengthX, int segLengthY){
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < hightY){
		for (int i=0; i<hightY; i++){
			dev_table[pos] = table[idx];
			pos += rowsize;
			idx += hightY;
		}
	}
}

__device__ void flagRead(int curBatch, volatile int *dev_lock, int thread, int idx, int YoverX, int xseg){
	if (thread == 0){
		int limit = min(idx+YoverX, xseg);
		
		printf("curBatch: %d, tile: %d, limit: %d, dev_lock[curBatch]: %d\n", curBatch, idx, limit, dev_lock[curBatch]);

	 	while(dev_lock[curBatch] < limit){
		}

		printf("curBatch: %d, tile: %d, is permit to proceed, dev_lock[curBatch]: %d\n", curBatch, idx, dev_lock[curBatch]);
	}
	__syncthreads();
}

__device__ void flagWrite(int curBatch, volatile int *dev_lock, int thread){
	if (thread == 0){
		dev_lock[curBatch+1] += 1;
	}
	__syncthreads();
}

__global__ void GPU(volatile int *dev_table, int *dev_arr1, int *dev_arr2, volatile int *dev_lock, int curBatch, int curStartAddress, int rowtiles, int hightY, int resX, int tileX, int tileY, int paddX, int paddY, int rowStartOffset, int rowsize, int colsize, int xseg, int yseg, int YoverX, int n1, int n2){ 
	//We assume row size n2 is the multiple of 32 and can be completely divided by tileX.
	//on K40, tile size is max to 48K, which is 128*96; on pascal and volta, tile size is max to 64K which is 128*128
	//This code, length of x axis cannot be larger than y axis for each tile.
	//For each row, the first tile and the last tile are computed separately from the other tiles.
	//No padding added, thus the first tile of each row and the first row requires statement check to set dependency to 0 for the edge elements.
	//size of the shared memory is determined by the GPU architecture.
	
	__shared__ int table[12288];

	int thread = threadIdx.x;
	int tileStartOffset, lvlStartAddress; 
	int glbStartX;
	int segLengthX = tileX + paddX;
	int segLengthY = tileY + paddY;
	int arrX = 0;
	int arrY = curBatch * tileY;
	int idxx, idxy, idx;
	int tile = 1;

//***********************************************************************************************************************************
	//processing the first tile of the row
	glbStartX = curStartAddress;
	flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
	
	if (thread == 0 && curBatch == row){
		printf("Before computation. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
		for(int i=0; i<colsize; i++){
			for(int j=0; j<rowsize; j++){	
				printf("%d ", dev_table[i*rowsize+j]);
			}
			printf("\n");
		}
		printf("\n");
	}


	moveToShareRec(&table[0], dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);				
	__syncthreads();
	__threadfence_block();
	
	if (thread == 0 && curBatch == row){	
		for (int i=0; i<segLengthY; i++){
			for (int j = 0; j<segLengthX; j++){
				printf("%d ", table[i*segLengthX+j]);
			}
			printf("\n");
		}
		printf("\n");
	}

	//first tile is irregular, concurrency is changed from 1 to hightY
	//the x length and y length of the first tile and the last tile are equal.
	tileStartOffset = paddY * segLengthX + paddX;
	for (int i=0; i<hightY; i++){
		lvlStartAddress = tileStartOffset + i;
		
		if(thread <= i){
			idx = lvlStartAddress + segLengthX * thread - thread;
			idxx = arrX + (i - thread);
			idxy = arrY + thread;
			table[idx] = max(table[idx-1], table[idx-segLengthX]);
			if(dev_arr1[idxx] == dev_arr2[idxy]){
				table[idx] = table[idx-segLengthX-1] + 1;	
			}	
//			printf("curBatch: %d, tile: %d, thread: %d, idx: %d, idxx: %d, x[idxx]: %d, idxy: %d, y[idxy]: %d, table[idx]: %d\n", curBatch, tile, thread, idx, idxx, dev_arr1[idxx], idxy, dev_arr2[idxy], table[idx]);						
		}
		__syncthreads();
	}

	
	if (thread == 0 && curBatch==row){	
		printf("After computation, in shared memory\n");
		for (int i=0; i<segLengthY; i++){
			for (int j = 0; j<segLengthX; j++){
				printf("%d ", table[i*segLengthX+j]);
			}
			printf("\n");
		}
		printf("\n");
	}
	
	moveToGlobalRec(&table[0], dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);				
	__threadfence();
	__syncthreads();

	if (thread == 0 && curBatch == row){
		printf("After computation. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
		for(int i=0; i<colsize; i++){
			for(int j=0; j<rowsize; j++){	
				printf("%d ", dev_table[i*rowsize+j]);
			}
			printf("\n");
		}
		printf("\n");
	}
	
	//update the tile beginning pos for the next tile.
	glbStartX += segLengthY;
	arrX = tileY;
	flagWrite(curBatch, dev_lock, thread);
//***********************************************************************************************************************************
	//hyperlane tiles, assume all hyperlane tiles have segLengthx == tileX
	tileStartOffset = paddY * segLengthX + paddX;
	for (tile = 2; tile < xseg; tile++){
		flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
	
//		printf("curBatch: %d, tile: %d, thread: %d is permit to read data from global.\n", curBatch, tile, thread);
		
		if (thread == 0 && curBatch == row){
			printf("Before computation. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			for(int i=0; i<colsize; i++){
				for(int j=0; j<rowsize; j++){	
					printf("%d ", dev_table[i*rowsize+j]);
				}
				printf("\n");
			}
			printf("\n");
		}
	
	
		moveToShare(table, dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);
		__syncthreads();
		__threadfence_block();


		if (thread == 0 && curBatch == row){	
			for (int i=0; i<segLengthY; i++){
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[i*segLengthX+j]);
				}
				printf("\n");
			}
			printf("\n");
		}
		
		lvlStartAddress = tileStartOffset;
		if (thread < tileY){
			for (int i=0; i<tileX; i++){
				idx = lvlStartAddress + thread * segLengthX;
				idxx = arrX + (i - thread);
				idxy = arrY + thread;
				table[idx] = max(table[idx-1], table[idx-segLengthX-1]);
				if(dev_arr1[idxx] == dev_arr2[idxy]){
					table[idx] = table[idx-segLengthY-2] + 1;
				}	
 			
//			printf("curBatch: %d, thread: %d, idx: %d, idxx: %d, idxy: %d, table[idx]: %d\n", curBatch, thread, idx, idxx, idxy, table[idx]);						
				lvlStartAddress += 1;
				__syncthreads();
			}
		}
		//need modification, only copy the new updated elements back to the global memory. Also modify moveToGlobalRec
		moveToGlobal(table, dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);
		
		__threadfence();
		__syncthreads();
		//update the tile beginning pos for the next tile.
		glbStartX += tileX;
		arrX+=tileX;

	if (thread == 0 && curBatch == row){	
		printf("After computation, in shared memory\n");
		for (int i=0; i<segLengthY; i++){
			for (int j = 0; j<segLengthX; j++){
				printf("%d ", table[i*segLengthX+j]);
			}
			printf("\n");
		}
		printf("\n");
	}

		if (thread == 0 && curBatch == row){
			printf("After computation. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
			for(int i=0; i<colsize; i++){
				for(int j=0; j<rowsize; j++){	
					printf("%d ", dev_table[i*rowsize+j]);
				}
				printf("\n");
			}
			printf("\n");
		}

		flagWrite(curBatch, dev_lock, thread);
	}

//************************************************************************************************************************************
	//the last tile, which is a half of the rectangular
	flagRead(curBatch, dev_lock, thread, xseg, YoverX, xseg);
	
	if (thread == 0 && curBatch == row){
		printf("Before computation. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
		for(int i=0; i<colsize; i++){
			for(int j=0; j<rowsize; j++){	
				printf("%d ", dev_table[i*rowsize+j]);
			}
			printf("\n");
		}
		printf("\n");
	}
	glbStartX += (rowsize + tileX - hightY);

	moveToShareLast(&table[0], dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);				
	__syncthreads();
	__threadfence_block();

		if (thread == 0 && curBatch == row){
			printf("last tile glbStartX: %d, rowsize: %d, hightY: %d\n", glbStartX, rowsize, hightY);	
			for (int i=0; i<hightY; i++){
				for (int j = 0; j<hightY; j++){
					printf("%d ", table[i*hightY+j]);
				}
				printf("\n");
			}
			printf("\n");
		}

	//last tile is irregular, concurrency is changed from hightY-1 to 1
	//the x length and y length of the first tile and the last tile are equal.
	tileStartOffset = hightY;
	for (int i=hightY-1; i>0; i--){
		lvlStartAddress = tileStartOffset + i;
		
		if(thread < i){
			idx = lvlStartAddress + segLengthX * thread - thread;
			idxx = n1 - thread;
			idxy = (arrY + 1) + thread;
			table[idx] = max(table[idx-1], table[idx-segLengthX]);
			if(dev_arr1[idxx] == dev_arr2[idxy]){
				table[idx] = table[idx-segLengthX-1] + 1;	
			}							
		}
		tileStartOffset += hightY;
		__syncthreads();
	}
	
	moveToGlobalLast(&table[0], dev_table, glbStartX, thread, tileX, rowsize, hightY, segLengthX, segLengthY);				
	__syncthreads();	
	
	if (thread == 0 && curBatch == row){	
		printf("After computation, in shared memory\n");
		for (int i=0; i<hightY; i++){
			for (int j = 0; j<hightY; j++){
				printf("%d ", table[i*hightY+j]);
			}
			printf("\n");
		}
		printf("\n");
	}

		if (thread == 0 && curBatch == row){
			printf("After computation. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
			for(int i=0; i<colsize; i++){
				for(int j=0; j<rowsize; j++){	
					printf("%d ", dev_table[i*rowsize+j]);
				}
				printf("\n");
			}
			printf("\n");
		}

	flagWrite(curBatch, dev_lock, thread);
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

int LCS(int n1, int n2, int *arr1, int *arr2){
	int lcslength;
	int paddX = 3, paddY = 1;

	//tileY must be larger than tileX
	int tileX = 2;
	int tileY = 2;
	int rowsize = paddX + n2;
	int colsize = paddY + n1;

	int *dev_arr1, *dev_arr2;
	int *dev_table, *dev_lock;

	int *table;

	table = new int[colsize * rowsize];

	size_t freeMem, totalMem;

	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_table, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMalloc(&dev_arr1, n1*sizeof(int));
	hipMalloc(&dev_arr2, n2*sizeof(int));

	hipMemset(dev_table, 0, tablesize * sizeof(int));
	hipMemcpy(dev_arr1, arr1, n1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr2, arr2, n2*sizeof(int), hipMemcpyHostToDevice);

	int threadPerBlock = max(tileY + paddY, tileX + paddX);
	int blockPerGrid = 1;
	int numStream = 15;

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	//For hyperlane tiles, if tileX!=tileY, the X length of the first tile and the last tile are equal to tileY.
//	int xseg = (n1+tileX-1) / tileX;
	int xseg = ((n1-tileY) + tileX - 1) / tileX + 2;
	int yseg = (n2+tileY-1) / tileY;

	hipMalloc(&dev_lock, (yseg+1) * sizeof(int));	
	hipMemset(dev_lock, 0, (yseg + 1) * sizeof(int));
	hipMemset(dev_lock, xseg+1, sizeof(int));
	hipStream_t stream[numStream];
	for (int s=0; s<numStream; s++)
		hipStreamCreate(&stream[s]);
	
	//instead of calling kernels along anti-diagonals, we now schedule kernels for each row.
	//We assume X axis is completly divided by tileX
	for(int curBatch = 0; curBatch < yseg; curBatch++){
		int curSMStream = curBatch % yseg;
		int resY = n1 - curBatch * tileY;
		int hightY = min(tileY, resY);
		int resX = (n2 - tileY) % tileX;
		int curStartAddress = curBatch * tileY * rowsize;
		int rowStartOffset = paddY * rowsize + paddX;
		int rowtiles = xseg + 1;
	
		GPU<<<blockPerGrid, threadPerBlock, 0, stream[curSMStream]>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, hightY, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
	}
	
	hipMemcpy(&lcslength,&dev_table[tablesize-1], sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(table, dev_table, colsize*rowsize*sizeof(int), hipMemcpyDeviceToHost);
/*
	//display table
	cout << "full table: " << endl;
	for (int i=0; i<colsize; i++){
		for (int j=0; j<rowsize; j++){
			cout << table[i * rowsize + j] << " ";
		}
		cout << endl;
	}
*/	

	for (int s=0; s<numStream; s++)
		hipStreamDestroy(stream[s]);
	
	hipFree(dev_arr1);
	hipFree(dev_arr2);
	hipFree(dev_table);
	hipFree(dev_lock);
	delete[] table;

	return lcslength;
}

