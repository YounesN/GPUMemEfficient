
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

//#define ALL
//#define DEBUG1
//#define DEBUG2
//#define DEBUG3

using namespace std;
__device__ int row = 0;

__device__ void moveToShare(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int segLengthX, int segLengthY, int paddX){
	//potential bank conflict for accessing the data of each anti-diagonal
	//to avoid bank conflict, have to adjust the memory layout by introducing dummy elements.
	//padding elements can be used as the dummy elements, but the number of padding of each dimension has to be an odd number.
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
//	if (thread < paddX){
		for (int i=0; i<segLengthY; i++){
//			printf("thread: %d, segLengthX: %d, pos: %d, idx: %d\n", thread, segLengthX, pos, idx);			
			
			table[idx] = dev_table[pos];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
}

__device__ void moveToGlobal(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int segLengthX, int segLengthY){
	int pos = tileAddress + thread;
	int idx = thread;
	//If y dimension cannot be completely divided by tileY, this code causes errors.
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			dev_table[pos] = table[idx];
			pos += (rowsize - 1);
			idx += segLengthX;
		}	
	}
}

__device__ void moveToShareRec(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int segLengthX, int segLengthY){
	//This function is designed for the first and the last tiles, which can be treate as rectangular but not hyperlane.
	//Rectangular tile does not have bank conflict issue.
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			table[idx] = dev_table[pos];
			pos += rowsize;
			idx += segLengthX;
		}	
	}
}

__device__ void moveToGlobalRec(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int segLengthX, int segLengthY){
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			dev_table[pos] = table[idx];
			pos += rowsize;
			idx += segLengthX;
		}
	}	
}

__device__ void moveToShareLast(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int segLengthX, int segLengthY){
	//This function is designed for the last tiles, which can be treate as rectangular but not hyperlane.
	//Rectangular tile does not have bank conflict issue.
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			table[idx] = dev_table[pos];
			pos += rowsize;
			idx += segLengthX;
		}	
	}
}

__device__ void moveToGlobalLast(volatile int *table, volatile int *dev_table, int tileAddress, int thread, int tileX, int rowsize, int segLengthX, int segLengthY){
	int pos = tileAddress + thread;
	int idx = thread;
	if (thread < segLengthX){
		for (int i=0; i<segLengthY; i++){
			dev_table[pos] = table[idx];
			pos += rowsize;
			idx += segLengthX;
		}
	}
}

__device__ void flagRead(int curBatch, volatile int *dev_lock, int thread, int idx, int YoverX, int xseg){
	if (thread == 0){
		int limit = min(idx+YoverX, xseg);
/*
#ifdef DEBUG		
		printf("curBatch: %d, tile: %d, limit: %d, dev_lock[curBatch]: %d\n", curBatch, idx, limit, dev_lock[curBatch]);
#endif
*/
	 	while(dev_lock[curBatch] < limit){
		}
/*
#ifdef DEBUG
		printf("curBatch: %d, tile: %d, is permit to proceed, dev_lock[curBatch]: %d\n", curBatch, idx, dev_lock[curBatch]);
#endif
*/
	}
	__syncthreads();
}

__device__ void flagWrite(int curBatch, volatile int *dev_lock, int thread){
	if (thread == 0){
		dev_lock[curBatch+1] += 1;
	}
	__syncthreads();
}

__global__ void GPU(volatile int *dev_table, int *dev_arr1, int *dev_arr2, volatile int *dev_lock, int curBatch, int curStartAddress, int rowtiles, int resX, int tileX, int tileY, int paddX, int paddY, int rowStartOffset, int rowsize, int colsize, int xseg, int yseg, int YoverX, int n1, int n2){ 
	//We assume row size n2 is the multiple of 32 and can be completely divided by tileX.
	//on K40, tile size is max to 48K, which is 128*96; on pascal and volta, tile size is max to 64K which is 128*128
	//This code, length of x axis cannot be larger than y axis for each tile.
	//For each row, the first tile and the last tile are computed separately from the other tiles.
	//No padding added, thus the first tile of each row and the first row requires statement check to set dependency to 0 for the edge elements.
	//size of the shared memory is determined by the GPU architecture.
	
#ifdef DEBUG
	if (threadIdx.x == 0){
		printf("This is curBatch: %d, curStartAddress: %d\n", curBatch, curStartAddress);
	}
	__syncthreads();
#endif

	volatile __shared__ int table[12288];

	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int tileStartOffset, lvlStartAddress; 
	int glbStartX;
	int segLengthX = tileX + paddX;
	int segLengthY = tileY + paddY;
	int arrX = 0;
	int arrY = curBatch * tileY;
	int idxx, idxy, idx;
	int tile = 1;

//***********************************************************************************************************************************
	//processing the first tile of the row
	glbStartX = curStartAddress;
	flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
	
	int highY = tileY;
	int piece = tileY / tileX;

	for (int p = 0; p < piece; p++){

#ifdef DEBUG1
#ifdef ALL
		if (thread == 32 && curBatch == row ){	
#endif
#ifndef ALL
		if (thread == 32){
#endif
			printf("Before move data share memory. curBatch: %d, tile: %d, p: %d, xseg: %d, glbStartX: %d, segLengthX: %d, segLengthY: %d\n", curBatch, tile, p, xseg, glbStartX, segLengthX, segLengthY);
			//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[highY *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
		__syncthreads();
#endif		
		moveToShareRec(&table[0], dev_table, glbStartX, thread, tileX, rowsize, segLengthX, segLengthY);				
		__syncthreads();
		__threadfence_system();
#ifdef DEBUG1
#ifdef ALL	
		if (thread == 32 && curBatch == row){	
#endif
#ifndef ALL
		if (thread == 32){
#endif	
			printf("Before computation, share memory. curBatch: %d, tile: %d, p: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, p, xseg, glbStartX);
			//for (int i=0; i<segLengthY; i++){

			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[highY *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
		__syncthreads();
#endif
		//first tile is irregular, concurrency is changed from 1 to hightY
		//the x length and y length of the first tile and the last tile are equal.
		tileStartOffset = paddY * segLengthX + paddX;
/*		for (int i=0; i<segLengthY; i++){
			lvlStartAddress = tileStartOffset + i;
		
			if(thread <= i){
				idx = lvlStartAddress + segLengthX * thread - thread;
				idxx = arrX + (i - thread);
				idxy = arrY + thread;
				table[idx] = max(table[idx-1], table[idx-segLengthX]);
				if(dev_arr1[idxx] == dev_arr2[idxy]){
					table[idx] = table[idx-segLengthX-1] + 1;	
				}	
			}
			__syncthreads();
		}
*/
		//length Y > length X, diagonal first element starts from Y axis instead of X axis for calculating the address.
		int concurrency;
		for (int i=0; i<highY; i++){
			lvlStartAddress = tileStartOffset + i * segLengthX;
			concurrency = min(tileX, i);
			if(thread <= concurrency){
				idx = lvlStartAddress - segLengthX * thread + thread;
				idxx = arrX + thread;
				idxy = arrY + i - thread;
				table[idx] = max(table[idx-1], table[idx-segLengthX]);
				if(dev_arr1[idxx] == dev_arr2[idxy]){
					table[idx] = table[idx-segLengthX-1] + 1;	
				}	
	//			printf("curBatch: %d, tile: %d, thread: %d, idx: %d, idxx: %d, x[idxx]: %d, idxy: %d, y[idxy]: %d, table[idx]: %d\n", curBatch, tile, thread, idx, idxx, dev_arr1[idxx], idxy, dev_arr2[idxy], table[idx]);						
			}
			__syncthreads();
		}
#ifdef DEBUG1
#ifdef ALL	
		if (thread == 32 && curBatch==row){	
#endif
#ifndef ALL
		if (thread == 32){
#endif
			printf("After computation, in shared memory\n");
			//for (int i=0; i<segLengthY; i++){
				{	
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[highY *segLengthX+j]);
					}
					printf("\n");
					//for (int j = 0; j<segLengthX; j++){
					//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
					//}
					//printf("\n");
				}
			printf("\n");
		}
		__syncthreads();
#endif

		moveToGlobalRec(&table[0], dev_table, glbStartX, thread, tileX, rowsize, segLengthX, segLengthY);				
		__threadfence_system();
		__syncthreads();


#ifdef DEBUG
#ifdef ALL
		if (thread == 32 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 32){
#endif
			printf("After computation in global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + highY *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
//			}
			printf("\n");
		}
		__syncthreads();
#endif
		arrX += tileX;		
		glbStartX += tileX;
		highY -= tileX;
		__syncthreads();
	}
	//update the tile beginning pos for the next tile.
//	glbStartX += (tileY + 1);
	glbStartX += 1;
	arrX = tileY;
	flagWrite(curBatch, dev_lock, thread);
//***********************************************************************************************************************************
	//hyperlane tiles
	tileStartOffset = paddY * segLengthX + paddX;
	for (tile = 2; tile < xseg; tile++){
		flagRead(curBatch, dev_lock, thread, tile, YoverX, xseg);
	
//		printf("curBatch: %d, tile: %d, thread: %d is permit to read data from global.\n", curBatch, tile, thread);

#ifdef DEBUG2
#ifdef ALL		
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("Before computation global memory. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j - 1]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j - (segLengthY-1)]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j - i * (segLengthY-1)]);
				//}
				//printf("\n");
//			}
			printf("\n");
		}
		__syncthreads();
#endif	

#ifdef DEBUG2
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3 ){	
#endif
#ifndef ALL
		if (thread == 0){
#endif
			printf("Before move data to share memory. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
		__syncthreads();
#endif		
		moveToShare(&table[0], dev_table, glbStartX, thread, tileX, rowsize, segLengthX, segLengthY, paddX);
		__syncthreads();
		__threadfence_system();
/*
#ifdef DEBUG2
#ifdef ALL		
		if (thread == 0 && curBatch == row && tile <=3){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("Before computation global. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + i*rowsize+j - i]);
				}
				printf("\n");
			}
			printf("\n");
		}
#endif
*/	

#ifdef DEBUG2
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3 ){	
#endif
#ifndef ALL
		if (thread == 0){
#endif
			printf("Before computation share. curBatch: %d, tile: %d, xseg: %d, glbStartX: %d\n", curBatch, tile, xseg, glbStartX);
			//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-tileX) *segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
			printf("\n");
		}
		__syncthreads();
#endif		

		lvlStartAddress = tileStartOffset;
		for (int i=0; i<tileX; i++){
//this is expensive especially when tileX is large. However, if we put if statement outside the loop, we face syncthreads issue.
//New feature warp level syncronize or thread group syncronize may solve the problem.
			if (thread < tileY){
				idx = lvlStartAddress + thread * segLengthX;
				idxx = arrX + (i - thread);
				idxy = arrY + thread;
				table[idx] = max(table[idx-1], table[idx-segLengthX-1]);
				if(dev_arr1[idxx] == dev_arr2[idxy]){
					table[idx] = table[idx-segLengthX-2] + 1;
				}	
				lvlStartAddress += 1;
			}
			__syncthreads();
		}

#ifdef DEBUG2
#ifdef ALL
	if (thread == 0 && curBatch == row && tile<=3){	
#endif
#ifndef ALL
	if (thread == 0){
#endif
		printf("After computation, in shared memory.\n");
		//for (int i=0; i<segLengthY; i++){
			{	
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-tileX) * segLengthX+j]);
				}
				printf("\n");
				for (int j = 0; j<segLengthX; j++){
					printf("%d ", table[(segLengthY-1) *segLengthX+j]);
				}
				printf("\n");
				//for (int j = 0; j<segLengthX; j++){
				//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
				//}
				//printf("\n");
			}
		printf("\n");
	}
	__syncthreads();
#endif

		//need modification, only copy the new updated elements back to the global memory. Also modify moveToGlobalRec
		moveToGlobal(&table[0], dev_table, glbStartX, thread, tileX, rowsize, segLengthX, segLengthY);
//		moveToGlobal(&table[paddX], dev_table, glbStartX + paddX, thread, tileX, rowsize, segLengthX, segLengthY);
		
		__threadfence_system();
		__syncthreads();

#ifdef DEBUG2
#ifdef ALL
		if (thread == 0 && curBatch == row && tile <= 3){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation, global memory. curBatch: %d, tile: %d, glbStartX: %d\n", curBatch, tile, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + rowsize+j - 1]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-tileX) *rowsize+j - (segLengthY-tileX-1)]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j - (segLengthY-1)]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j - i * (segLengthY-1)]);
				//}
				//printf("\n");
		//	}
			printf("\n");
		}
		__syncthreads();
#endif

		//update the tile beginning pos for the next tile.
		glbStartX += tileX;
		arrX+=tileX;
		flagWrite(curBatch, dev_lock, thread);
	}

//************************************************************************************************************************************
	//the last tile, which is a half of the rectangular
	flagRead(curBatch, dev_lock, thread, xseg, YoverX, xseg);
	glbStartX = curStartAddress + rowsize - tileY - paddX;
	
	piece = tileY / tileX;
	highY = tileX;
	
	for (int p=0; p<piece; p++){

#ifdef DEBUG3
#ifdef ALL	
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif
			printf("Before computation global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY - 1) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY ) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY + 1) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i* (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
			//}
			printf("\n");
		}
		__syncthreads();
#endif
	
		moveToShareLast(&table[0], dev_table, glbStartX, thread, tileX, rowsize, segLengthX, segLengthY);				
		__syncthreads();
		__threadfence_block();

#ifdef DEBUG3
#ifdef ALL
			if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
			if (thread == 0){
#endif
				printf("last tile share memory before computation, glbStartX: %d, p: %d, rowsize: %d, segLengthY: %d\n", glbStartX, p, rowsize, segLengthY);	
			//	for (int i=0; i<segLengthY; i++){
				{	
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-highY - 1) * segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY + 1) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-1) *segLengthX+j]);
					}
					printf("\n");
					//for (int j = 0; j<segLengthX; j++){
					//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
					//}
					//printf("\n");
				}
				printf("\n");
			}
			__syncthreads();
#endif

		//last tile is irregular, concurrency is changed from hightY-1 to 1
		//the x length and y length of the first tile and the last tile are equal.
		int concurrency;
		tileStartOffset = segLengthX * (segLengthY - highY);
		for (int i=highY; i>0; i--){
			lvlStartAddress = tileStartOffset + segLengthX - 1;
			concurrency = min(tileX, i);	
			if(thread < concurrency){
				idx = lvlStartAddress + segLengthX * thread - thread;
				idxx = (n1 - tileY + highY - 1) - thread;
				//idxy = (n2 - i) + thread;
				idxy = (arrY + tileY - i) + thread;
				table[idx] = max(table[idx-1], table[idx-segLengthX]);
				if(dev_arr1[idxx] == dev_arr2[idxy]){
					table[idx] = table[idx-segLengthX-1] + 1;	
				}							
			}
			tileStartOffset += segLengthX;
			__syncthreads();
		}
	
		moveToGlobalLast(&table[0], dev_table, glbStartX, thread, tileX, rowsize, segLengthX, segLengthY);				
		__syncthreads();	

#ifdef DEBUG3
#ifdef ALL	
		if (thread == 0 && curBatch == row){	
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation, in shared memory\n");
			//for (int i=0; i<segLengthY; i++){
				{	
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-highY - 1) * segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY- highY + 1) *segLengthX+j]);
					}
					printf("\n");
					for (int j = 0; j<segLengthX; j++){
						printf("%d ", table[(segLengthY-1) *segLengthX+j]);
					}
					printf("\n");
					//for (int j = 0; j<segLengthX; j++){
					//	printf("%d ", table[i * (segLengthY-1) *segLengthX+j]);
					//}
					//printf("\n");
				}
			printf("\n");
		}
		__syncthreads();
#endif


#ifdef DEBUG3
#ifdef ALL
		if (thread == 0 && curBatch == row){
#endif
#ifndef ALL
		if (thread == 0){
#endif	
			printf("After computation. global memory. curBatch: %d, tile: %d, p: %d, glbStartX: %d\n", curBatch, tile, p, glbStartX);
			//for(int i=0; i<segLengthY; i++){
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY - 1) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY ) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY - highY + 1) * rowsize+j]);
				}
				printf("\n");
				for(int j=0; j<segLengthX; j++){	
					printf("%d ", dev_table[glbStartX + (segLengthY-1) *rowsize+j]);
				}
				printf("\n");
				//for(int j=0; j<segLengthX; j++){	
				//	printf("%d ", dev_table[glbStartX + i * (segLengthY-1) *rowsize+j]);
				//}
				//printf("\n");
			//}
			printf("\n");
		}
		__syncthreads();
#endif
		glbStartX += tileX;
		highY += tileX;
	}
	
	flagWrite(curBatch, dev_lock, thread);
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

int LCS(int n1, int n2, int *arr1, int *arr2, int paddX, int paddY, int *table){
	int lcslength;

	//tileY must be larger than tileX
	int tileX = 64;
	int tileY = 128;
	int rowsize = paddX + n2;
	int colsize = paddY + n1;

	int *dev_arr1, *dev_arr2;
	volatile int *dev_table, *dev_lock;
	int *lock;
	size_t freeMem, totalMem;
	
	
	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_table, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMalloc(&dev_arr1, n1*sizeof(int));
	hipMalloc(&dev_arr2, n2*sizeof(int));

	hipMemset((void*)dev_table, 0, tablesize * sizeof(int));
	hipMemcpy(dev_arr1, arr1, n1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr2, arr2, n2*sizeof(int), hipMemcpyHostToDevice);

	int threadPerBlock = max(tileY + 32, tileX + 32);
	int blockPerGrid = 1;
	int numStream = 15;

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	//For hyperlane tiles, if tileX!=tileY, the X length of the first tile and the last tile are equal to tileY.
//	int xseg = (n1+tileX-1) / tileX;
	int xseg = ((n1-tileY) + tileX - 1) / tileX + 2;
	int yseg = (n2+tileY-1) / tileY;

	lock = new int[yseg+1];
	lock[0] = xseg+1;
	for (int i=1; i<yseg+1; i++)
		lock[i] = 0;
	hipMalloc(&dev_lock, (yseg+1) * sizeof(int));	
//	cudaMemset((void*)dev_lock, 0, (yseg + 1) * sizeof(int));
//	cudaMemset((void*)dev_lock, xseg+1, sizeof(int));
	hipMemcpy((void*)dev_lock, lock, (yseg+1)*sizeof(int), hipMemcpyHostToDevice);
	hipStream_t stream[numStream];
	for (int s=0; s<numStream; s++)
		hipStreamCreate(&stream[s]);
	
	//instead of calling kernels along anti-diagonals, we now schedule kernels for each row.
	//We assume X axis is completly divided by tileX
	for(int curBatch = 0; curBatch < yseg; curBatch++){
		int curSMStream = curBatch % numStream;
		//int resY = n1 - curBatch * tileY;
		int resX = (n2 - tileY) % tileX;
		int curStartAddress = curBatch * tileY * rowsize;
		int rowStartOffset = paddY * rowsize + paddX;
		int rowtiles = xseg + 1;
//		cout << endl << "curBatch: " << curBatch << ", yseg: " << yseg << endl;	
		GPU<<<blockPerGrid, threadPerBlock, 0, stream[curSMStream]>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
//		GPU<<<blockPerGrid, threadPerBlock>>>(dev_table, dev_arr1, dev_arr2, dev_lock, curBatch, curStartAddress, rowtiles, resX, tileX, tileY,  paddX, paddY, rowStartOffset, rowsize, colsize, xseg, yseg, tileY/tileX, n1, n2);			
//		cudaDeviceSynchronize();
	}
	hipMemcpy(&lcslength, (void*)&dev_table[tablesize-1], sizeof(int), hipMemcpyDeviceToHost);
	//cudaMemcpy(table, (void*)dev_table, tablesize*sizeof(int), cudaMemcpyDeviceToHost);

#ifdef DEBUG
	//display table
	cout << "grid size: " << blockPerGrid << ", block size: " << threadPerBlock << ", full table: " << endl;
	for (int i=0; i<colsize; i++){
		for (int j=0; j<rowsize; j++){
			cout << table[i * rowsize + j] << " ";
		}
		cout << endl;
	}
	
#endif
	for (int s=0; s<numStream; s++)
		hipStreamDestroy(stream[s]);
	
	hipFree(dev_arr1);
	hipFree(dev_arr2);
	hipFree((void*)dev_table);
	hipFree((void*)dev_lock);
	delete[] lock;

	return lcslength;
}

